#include "hip/hip_runtime.h"
// This is the real Hello World for CUDA!
//It takes the string 'Hello', prints it, then passes it to cuda with an array of offsets
// Then the offsets are added in parallel to produce the string world! 
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ros/ros.h"
#include "sensor_msgs/PointCloud2.h"
#include <sensor_msgs/PointCloud.h>
#include <sensor_msgs/point_cloud_conversion.h>
#include <time.h>
#include <sensor_msgs/point_cloud2_iterator.h>
#include <string>
#include <cmath> 
#include "math.h"
#include <cstdlib>
#include <stdio.h>
#include <map>
#include <iterator>
#include <boost/lexical_cast.hpp>
#include <tf/transform_datatypes.h>
#include <nav_msgs/Odometry.h>
#include <geometry_msgs/Vector3.h>
#include "geometry_msgs/Quaternion.h"

// =========================================================================================================================
// This class file is to transport into include directory after development 


class Key3D{
	
private: 
	float x; 
	float y; 
	float z; 

public: 
	//Null constructor
	__host__ __device__ Key3D(){} 

	//preferred constructor
	__host__ __device__ Key3D(float x_in, float y_in, float z_in){
		x = x_in; 
		y = y_in; 
		z = z_in;  
	}

	//retrieve values for testing
	__host__ __device__ const float& getX() const {
		return x; 
	}
	__host__ __device__ const float& getY() const {
		return y; 
	}
	__host__ __device__ const float& getZ() const {
		return z; 
	}

	//the sorting operator for a map
	bool operator< (const Key3D& keyObj) const{

        if(keyObj.x < this->x){
			return true;
		} 
		return false;         
    }

	//destructor
	__host__ __device__ ~Key3D(){} 

}; 

const float epsilon = 0.0125; // epsilon to compare two floats. this value depends on the resolution we consider. (resolution /4 or lower )

struct VoxelKeyComparator
{	
	bool operator()(const Key3D &left, const Key3D &right) const{
		
		if((abs(left.getX()-right.getX()) > epsilon) || (abs(left.getY()-right.getY()) > epsilon) || (abs(left.getZ()-right.getZ()) > epsilon)){
			return (left.getX() <= right.getX());
			//return true; 
		}
		return false; 
	}
};

// This class is originally developed to generate within host. updating count parameter with device might require extra effort. 
class VoxelBody{		
private: 
	int point_count; 
	uint8_t r; 
	uint8_t g; 
	uint8_t b; 

public: 
	//Null constructor

	__host__ __device__ VoxelBody(){
		point_count = 0; 
		r = 0; 
		g = 0; 
		b = 0; 
	} 

	//full constructor
	__host__ __device__ VoxelBody(uint8_t r_in, uint8_t g_in, uint8_t b_in){
		point_count = 1; 	
		r = r_in; 
		g = g_in; 
		b = b_in; 						
	}
	// desctructor
	__host__ __device__ ~VoxelBody(){} 

	__host__ __device__ void updateVoxel(uint8_t r_in, uint8_t g_in, uint8_t b_in){
		point_count += 1; 
		if(r == 0 and g == 0 and b == 0){		// a previously mis initialized points
			r = r_in; 
			g = g_in; 
			b = b_in;
		}else{								// average the value within the voxel 
			r = (r + r_in)/2; 
			g = (g + g_in)/2; 
			b = (b + b_in)/2;
		}
	}

	__host__ __device__ const uint8_t& get_r() const { return r; }
	__host__ __device__ const uint8_t& get_g() const { return g; }
	__host__ __device__ const uint8_t& get_b() const { return b; }
	__host__ __device__ const int& get_point_count() const {return point_count; }
	
};

// ==============================================================================================================================

using namespace std; 

#define THREADS_PER_BLOCK 256		// the optimal value is number of cuda cores, if (#cuda cores < max th.pr.blck). 256 for TX2

const float resolution = 0.05f; 	// Resolution of 5 cm
const float half_resolution = resolution/2; // the half of the resolution. this value is used in calculations 

const float max_sensor_radius = 3.00f; 	// scan data further than this modulus value will not be considered. 
 

void serial_round_off_positional_coords(int size, float* x, float* y, float* z, float* x_result, float* y_result, float* z_result, 
	float x_trans, float y_trans, float z_trans, double sin_a, double sin_b, double sin_g, double cos_a, double cos_b, double cos_g){
	
	for(int i=0; i<size; i++){
		if( (abs(x[i]) < max_sensor_radius) and (abs(y[i]) < max_sensor_radius) and (abs(z[i]) < max_sensor_radius)){
					
			//B: Transformation model 1 -----------yaw only----------- for flat surface navigation-----------------------------------
			/*float x_temp = x[i]*cos_a - y[i]*sin_a + x_trans;  
			float y_temp = x[i]*sin_a + y[i]*cos_a + y_trans; 
			float z_temp = z[i] + z_trans; 
			
			x_result[i] = (ceilf(x_temp / resolution))*0.05 - half_resolution;
			y_result[i] = (ceilf(y_temp / resolution))*0.05 - half_resolution; 
			z_result[i] = (ceilf(z_temp / resolution))*0.05 - half_resolution; */
			//------------------------------------------------------------------------------------------------
			
			// C: Transformation model 2 ---------- Roll, pitch, yaw combined--------- for inclined planes navigation --------------
			float x_temp = x[i]*cos_a*cos_b + y[i]*cos_a*sin_b*sin_g - y[i]*sin_a*cos_g + z[i]*cos_a*sin_b*cos_g + z[i]*sin_a*sin_g + x_trans;  
			float y_temp = x[i]*sin_a*cos_b + y[i]*sin_a*sin_b*sin_g + y[i]*cos_a*cos_g + z[i]*sin_a*sin_b*cos_g - z[i]*cos_a*sin_g + y_trans; 
			float z_temp = x[i]*sin_b*-1 + y[i]*cos_b*sin_g + z[i]*cos_b*cos_g + z_trans; 
			
			x_result[i] = (ceil(x_temp / resolution))*0.05 - half_resolution;
			y_result[i] = (ceil(y_temp / resolution))*0.05 - half_resolution; 
			z_result[i] = (ceil(z_temp / resolution))*0.05 - half_resolution;
			// -----------------------------------------------------------------------------------------------------------------------------------------
			
		}else{
			x_result[i] = 0.00f; 
			y_result[i] = 0.00f; 
			z_result[i] = 0.00f; 

		} 
	}
	
}


void serial_remove_null_points(int camera_array_size, float* x, float* y, float* z, uint8_t* r, uint8_t* g, uint8_t* b, 
													float* x_cropped, float* y_cropped, float* z_cropped, uint8_t* r_cropped, uint8_t* g_cropped, uint8_t* b_cropped){
	// x, y, z are input arrays with rounded and range controlled poitn clouds
	// x_cropped etc. stores the new array with 0,0,0 points removed. 
	int index = 0; 
	for(int i=0; i<camera_array_size; i++){			// size of original set of point cloud
		if(x[i] != 0.00f){
			x_cropped[index] = x[i]; 
			y_cropped[index] = y[i]; 
			z_cropped[index] = z[i]; 
			r_cropped[index] = r[i]; 
			g_cropped[index] = g[i]; 
			b_cropped[index] = b[i]; 
			index +=1; 
		}
	}
}

static std::map<Key3D, VoxelBody, VoxelKeyComparator> voxel_map_downsampled; 

void serial_downsample_point_cloud(int array_size, float* x_cleaned, float* y_cleaned, float* z_cleaned, uint8_t* r_cleaned, uint8_t* g_cleaned, uint8_t* b_cleaned){

	Key3D temp_key; 
	VoxelBody temp_body; 
	
	for(int i=0; i<array_size; i++){
		temp_key = Key3D(x_cleaned[i], y_cleaned[i], z_cleaned[i]);
		temp_body = VoxelBody(r_cleaned[i], g_cleaned[i], b_cleaned[i]); 
		// Keep inerting for now. will genaralize rgb values and count later. 
		voxel_map_downsampled.insert(std::make_pair<Key3D, VoxelBody>(temp_key, temp_body));
	}
	
}

int downsampled_count = 0; 
void readings_function(int array_size, float* x_cleaned, float* y_cleaned, float* z_cleaned, uint8_t* r_cleaned, uint8_t* g_cleaned, uint8_t* b_cleaned, 
							std::map<Key3D, VoxelBody, VoxelKeyComparator> map_reading){

	Key3D temp_key; 
	VoxelBody temp_body; 
	
	for(int i=0; i<array_size; i++){
		temp_key = Key3D(x_cleaned[i], y_cleaned[i], z_cleaned[i]);
		temp_body = VoxelBody(r_cleaned[i], g_cleaned[i], b_cleaned[i]); 
		// Keep inerting for now. will genaralize rgb values and count later. 
		map_reading.insert(std::make_pair<Key3D, VoxelBody>(temp_key, temp_body));
	}
	downsampled_count =0; 
	
	std::map<Key3D, VoxelBody, VoxelKeyComparator>::iterator itr; 
	for(itr=map_reading.begin(); itr!=map_reading.end(); itr++){
		downsampled_count +=1;   
	}

}


int cudamain(sensor_msgs::PointCloud2 point_cloud_std, nav_msgs::Odometry odom_message_std, int size){ 
	//make_range_array(resolution, max_sensor_radius); 
	int array_size = size; 	 

	// convert quaternion orientation into roll, pith, yaw representation 
	double start_prep, end_prep; 
	start_prep = clock(); //----------------------------------------------------------------------
	double roll, pitch, yaw; 
	tf::Quaternion quat;
    tf::quaternionMsgToTF(odom_message_std.pose.pose.orientation, quat);	
    tf::Matrix3x3(quat).getRPY(roll, pitch, yaw);
	float x_position = (float) odom_message_std.pose.pose.position.x; 
	float y_position = (float) odom_message_std.pose.pose.position.y; 
	float z_position = (float) odom_message_std.pose.pose.position.z; 

	double sin_gamma = sin(roll); 
	double sin_beta = sin(pitch); 
	double sin_alpha = sin(yaw); 
	double cos_gamma = cos(roll); 
	double cos_beta = cos(pitch); 
	double cos_alpha = cos(yaw); 

	
	//std::cout<<"alpha: "<<yaw<<"  sin alpha: "<<sin_alpha<<std::endl;
	//std::cout<<"\n\n"; 

	int counter = 0; 
	int effective_point_count = 0; 
	//declare the arrray sets before reading the point cloud values 
	
	float *x, *y, *z; // for allocating position values of the points 
	float *x_rounded, *y_rounded, *z_rounded; 							// the intermediate results after rounding off the x, y, z, original values to the resolution 
	u_int8_t *r, *g, *b; // for color values of the point cloud 
	

	int size_position = array_size * sizeof(float);
	int size_color = array_size * sizeof(u_int8_t);
	
	x = (float *)malloc( size_position );
   	y = (float *)malloc( size_position );
	z = (float *)malloc( size_position );

	x_rounded = (float *)malloc( size_position );
   	y_rounded = (float *)malloc( size_position );
	z_rounded = (float *)malloc( size_position );
	
	r = (u_int8_t *)malloc( size_color );
    g = (u_int8_t *)malloc( size_color );
	b = (u_int8_t *)malloc( size_color );
	
	// positional data vector generation 
	for(sensor_msgs::PointCloud2ConstIterator<float> it(point_cloud_std, "x"); it!=it.end(); ++it){
		y[counter] = it[0] * -1; 
		z[counter] = it[1] * -1;
		x[counter] = it[2];
		counter+=1;  
		
	}
	counter = 0; 
    for(sensor_msgs::PointCloud2ConstIterator<u_int8_t> it_color(point_cloud_std, "rgb"); it_color!=it_color.end(); ++it_color){
		b[counter] = unsigned(it_color[0]);	
		g[counter] = unsigned(it_color[1]);	
		r[counter] = unsigned(it_color[2]); 
		counter+=1; 
	}
	counter = 0; 
	end_prep = clock(); //--------------------------------------------------------------------------------------------------------------
	double time_prep = (double)(end_prep - start_prep); 
	
	
	double start_round, end_round; 
	start_round = clock(); //-------------------------------------------------------------------------------------------------------------
	serial_round_off_positional_coords( array_size, x, y, z, x_rounded, y_rounded, z_rounded,
									x_position, y_position, z_position, sin_alpha, sin_beta, sin_gamma, cos_alpha, cos_beta, cos_gamma);
	end_round = clock();	//------------------------------------------------------------------------------------------------------------
	double time_round = (double)(end_round-start_round); 
	 
	
	for(int i=0; i<array_size; i++){		// counted the number of informative points 
		if(x_rounded[i] != 0.00f){
			effective_point_count += 1; 
		}
	}

	float *x_cleaned, *y_cleaned, *z_cleaned; 
	uint8_t *r_cleaned, *g_cleaned, *b_cleaned; 

	int size_cleaned = effective_point_count * sizeof(float);
	int size_colour_cleaned = effective_point_count * sizeof(uint8_t); 
	
	x_cleaned = (float *)malloc( size_cleaned );
	y_cleaned = (float *)malloc( size_cleaned );
	z_cleaned = (float *)malloc( size_cleaned );
	r_cleaned = (uint8_t *)malloc(size_colour_cleaned); 
	g_cleaned = (uint8_t *)malloc(size_colour_cleaned); 
	b_cleaned = (uint8_t *)malloc(size_colour_cleaned); 

	double start_null, end_null; 
	start_null = clock(); //--------------------------------------------------------------------------------------------------------------------------
	serial_remove_null_points(array_size, x_rounded, y_rounded, z_rounded, r, g, b, x_cleaned, y_cleaned, z_cleaned, r_cleaned, g_cleaned, b_cleaned);  
	end_null = clock();  // --------------------------------------------------------------------------------------------------------------------------
	double time_null = (double) (end_null - start_null); 
	
	double start_downsample, end_downsample; 
	start_downsample = clock(); //------------------------------------------------------------------------------------------------------------------------
	serial_downsample_point_cloud(effective_point_count, x_cleaned, y_cleaned, z_cleaned, r_cleaned, g_cleaned, b_cleaned); 
	
	end_downsample = clock(); //--------------------------------------------------------------------------------------
	double time_downsample = (double)(end_downsample-start_downsample); 
	
	std::string output_file_name = "result_map_key_generation.txt";
	ofstream offile;
	offile.open(output_file_name.c_str(), ios::trunc);
	if(offile.is_open()) { 
		std::map<Key3D, VoxelBody, VoxelKeyComparator>::iterator itr; 
		for(itr=voxel_map_downsampled.begin(); itr!=voxel_map_downsampled.end(); itr++){
			offile<<itr->first.getX()<<"\t"<<itr->first.getY()<<"\t"<<itr->first.getZ()<<"\t"<<unsigned(itr->second.get_r())<<"\t"<<unsigned(itr->second.get_g())<<"\t"<<unsigned(itr->second.get_b())<<std::endl;  
		}
		offile<<"\n\n"; 
		//std::cout<<"data written"<<std::endl; 											
								
	}
	offile.close();
	
	 
	std::cout<<time_prep<<"\t"<<time_round<<"\t"<<time_null<<"\t"<<time_downsample<<std::endl; 
	 
	free(x);
    free(y);
	free(z);
	free(r);
	free(g);
	free(b);
	free(x_rounded); 
	free(y_rounded); 
	free(z_rounded);
	
	free(x_cleaned); 
	free(y_cleaned); 
	free(z_cleaned); 
	free(r_cleaned); 
	free(g_cleaned); 
	free(b_cleaned); 

	return EXIT_SUCCESS; 	
}
