#include "hip/hip_runtime.h"
// this program will detect dynamic objects in the octomap 

#include <iostream>
#include <bits/stdc++.h> 
#include <stdint.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ros/ros.h"
#include "sensor_msgs/PointCloud2.h"
#include <sensor_msgs/PointCloud.h>
#include <sensor_msgs/point_cloud_conversion.h>
#include <time.h>
#include <sensor_msgs/point_cloud2_iterator.h>
#include <string>
#include <cmath> 
#include "math.h"
#include <cstdlib>
#include <stdio.h>
#include <map>
#include <iterator>
#include <boost/lexical_cast.hpp>
#include <tf/transform_datatypes.h>
#include <nav_msgs/Odometry.h>
#include <geometry_msgs/Vector3.h>
#include "geometry_msgs/Quaternion.h"


using namespace std; 

// =========================================================================================================================
// This class file is to transport into include directory after development 


float res_list_5cm[] = {3276.80, 1638.40, 819.20, 409.60, 204.80, 102.40, 51.20, 25.60, 12.80, 6.40, 3.20, 1.60, 0.80, 0.40, 0.20, 0.10, 0.05, 0.025}; // in meters

struct OctreeNode{
    //for now we consider a depth of 16 for data nodes. max reolution = 65536 cm. float is enough 
    float X; 
    float Y; 
    float Z; 
    uint8_t R; 
    uint8_t G; 
    uint8_t B; 
    //uint8_t level;          // possible replace this with 4 bit binary code, or not to have it at all   
    int8_t Occ;  
    OctreeNode *BNE; //hhl; 
    OctreeNode *BSE; //hll; 
    OctreeNode *BSW; //lll, 
    OctreeNode *BNW; //lhl; 
    OctreeNode *TNE; //hhh; 
    OctreeNode *TSE; //hlh; 
    OctreeNode *TSW; //llh, 
	OctreeNode *TNW; //lhh; 
	
	//constructor
	OctreeNode(){
		X = 0.00f; 
		Y = 0.00f; 
		Z = 0.00f; 
		R = 255;
		G = 255; 
		B = 255; 
		Occ = 0;
		BNE = NULL; 
		BSE = NULL; 
		BSW = NULL; 
		BNW = NULL; 
		TNE = NULL; 
		TSE = NULL; 
		TSW = NULL; 
		TNW = NULL;   
	}

};

OctreeNode *current;            // these are global varibale used to travel down the tree 
OctreeNode *parent; 
queue<OctreeNode*> leaf_nodes;

class Octree{

    OctreeNode *root; 

	void insertNode(float x, float y, float z, uint8_t r, uint8_t g, uint8_t b){
        // insert this node at the depth of 16. where all the leaf nodes are 
		if(root==NULL){
			return;
		}
		int depth = 0; //root level
        current = root; 
        //cout<<"starting from root"<<endl;  

		for(depth=0; depth<16; depth++){
			if(current->BNE == NULL){
				//cout<<depth<<" next level null"<<endl; 
				
				OctreeNode *node_BNE = new OctreeNode; 
				node_BNE->X = current->X + res_list_5cm[depth+2];
				node_BNE->Y = current->Y + res_list_5cm[depth+2]; 
				node_BNE->Z = current->Z - res_list_5cm[depth+2]; 
				current->BNE = node_BNE;
				//cout<<"established BNE node of level "<<depth<<endl; 
				
				OctreeNode *node_BSE = new OctreeNode; 
				node_BSE->X = current->X + res_list_5cm[depth+2];
				node_BSE->Y = current->Y - res_list_5cm[depth+2]; 
				node_BSE->Z = current->Z - res_list_5cm[depth+2];
				current->BSE = node_BSE;
				//cout<<"established BSE node of level "<<depth<<endl; 

				OctreeNode *node_BSW = new OctreeNode; 
				node_BSW->X = current->X - res_list_5cm[depth+2];
				node_BSW->Y = current->Y - res_list_5cm[depth+2]; 
				node_BSW->Z = current->Z - res_list_5cm[depth+2]; 
				current->BSW = node_BSW;
				//cout<<"established BSW node of level "<<depth<<endl;

				OctreeNode *node_BNW = new OctreeNode; 
				node_BNW->X = current->X - res_list_5cm[depth+2];
				node_BNW->Y = current->Y + res_list_5cm[depth+2]; 
				node_BNW->Z = current->Z - res_list_5cm[depth+2];
				current->BNW = node_BNW;
				//cout<<"established BNW node of level "<<depth<<endl;

				OctreeNode *node_TNE = new OctreeNode; 
				node_TNE->X = current->X + res_list_5cm[depth+2];
				node_TNE->Y = current->Y + res_list_5cm[depth+2]; 
				node_TNE->Z = current->Z + res_list_5cm[depth+2];
				current->TNE = node_TNE;
				//cout<<"established TNE node of level "<<depth<<endl; 
				
				OctreeNode *node_TSE = new OctreeNode; 
				node_TSE->X = current->X + res_list_5cm[depth+2];
				node_TSE->Y = current->Y - res_list_5cm[depth+2]; 
				node_TSE->Z = current->Z + res_list_5cm[depth+2];
				current->TSE = node_TSE;
				//cout<<"established TSE node of level "<<depth<<endl; 

				OctreeNode *node_TSW = new OctreeNode; 
				node_TSW->X = current->X - res_list_5cm[depth+2];
				node_TSW->Y = current->Y - res_list_5cm[depth+2]; 
				node_TSW->Z = current->Z + res_list_5cm[depth+2];
				current->TSW = node_TSW;
				//cout<<"established TSW node of level "<<depth<<endl;

				OctreeNode *node_TNW = new OctreeNode; 
				node_TNW->X = current->X - res_list_5cm[depth+2];
				node_TNW->Y = current->Y + res_list_5cm[depth+2]; 
				node_TNW->Z = current->Z + res_list_5cm[depth+2];
				current->TNW = node_TNW;
				//cout<<"established TNW node of level "<<depth<<endl;

				// maintain leaf nodes index. 
				if(depth == 15){
                    leaf_nodes.push(current->BNE);
                    leaf_nodes.push(current->BSE); 
                    leaf_nodes.push(current->BSW); 
                    leaf_nodes.push(current->BNW); 
                    leaf_nodes.push(current->TNE);
                    leaf_nodes.push(current->TSE); 
                    leaf_nodes.push(current->TSW); 
                    leaf_nodes.push(current->TNW);  
                }

			}else{
				//cout<<depth<<" next level not null"<<endl; 
				//cout<<"next level x  "<<current->BNE->X<<endl; 
			}

			if(x>current->X && y>current->Y && z<current->Z){
				current = current->BNE; 
			}else if(x>current->X && y<current->Y && z<current->Z){
				current = current->BSE; 
			}else if(x<current->X && y<current->Y && z<current->Z){
				current = current->BSW; 
			}else if(x<current->X && y>current->Y && z<current->Z){
				current = current->BNW; 
			}else if(x>current->X && y>current->Y && z>current->Z){
				current = current->TNE; 
			}else if(x>current->X && y<current->Y && z>current->Z){
				current = current->TSE; 
			}else if(x<current->X && y<current->Y && z>current->Z){
				current = current->TSW; 
			}else if(x<current->X && y>current->Y && z>current->Z){
				current = current->TNW; 
			}else{
				//std::cout<<"something else"<<endl; 
			}
		}

		// now we are at level 16. 
		// "current" pointer indicates to the exact node the voxel should be added. 
		if(current->Occ == 0){		// means not initialized
			current->Occ = 1; 
			current->R = r;
			current->G = g;
			current->B = b; 
		}else if(current->Occ == -1){	// we do not need to recognize this node as unknown 
			current->Occ = 1; 
			current->R = (current->R + r)/2;
			current->G = (current->G + g)/2;
			current->B = (current->B + b)/2;
		}else if(current->Occ <16){
			current->Occ += 1; 
			current->R = (current->R + r)/2;
			current->G = (current->G + g)/2;
			current->B = (current->B + b)/2;
		}else{
			// no case according to algorithm
		}
	}
	void insert_free_Node(float x, float y, float z){
        // insert this node at the depth of 16. where all the leaf nodes are 
		if(root==NULL){
			return;
		}
		int depth = 0; //root level
        current = root; 
        //cout<<"starting from root"<<endl;  

		for(depth=0; depth<16; depth++){
			if(current->BNE == NULL){
				//cout<<depth<<" next level null"<<endl; 
				
				OctreeNode *node_BNE = new OctreeNode; 
				node_BNE->X = current->X + res_list_5cm[depth+2];
				node_BNE->Y = current->Y + res_list_5cm[depth+2]; 
				node_BNE->Z = current->Z - res_list_5cm[depth+2]; 
				current->BNE = node_BNE;
				//cout<<"established BNE node of level "<<depth<<endl; 
				
				OctreeNode *node_BSE = new OctreeNode; 
				node_BSE->X = current->X + res_list_5cm[depth+2];
				node_BSE->Y = current->Y - res_list_5cm[depth+2]; 
				node_BSE->Z = current->Z - res_list_5cm[depth+2];
				current->BSE = node_BSE;
				//cout<<"established BSE node of level "<<depth<<endl; 

				OctreeNode *node_BSW = new OctreeNode; 
				node_BSW->X = current->X - res_list_5cm[depth+2];
				node_BSW->Y = current->Y - res_list_5cm[depth+2]; 
				node_BSW->Z = current->Z - res_list_5cm[depth+2]; 
				current->BSW = node_BSW;
				//cout<<"established BSW node of level "<<depth<<endl;

				OctreeNode *node_BNW = new OctreeNode; 
				node_BNW->X = current->X - res_list_5cm[depth+2];
				node_BNW->Y = current->Y + res_list_5cm[depth+2]; 
				node_BNW->Z = current->Z - res_list_5cm[depth+2];
				current->BNW = node_BNW;
				//cout<<"established BNW node of level "<<depth<<endl;

				OctreeNode *node_TNE = new OctreeNode; 
				node_TNE->X = current->X + res_list_5cm[depth+2];
				node_TNE->Y = current->Y + res_list_5cm[depth+2]; 
				node_TNE->Z = current->Z + res_list_5cm[depth+2];
				current->TNE = node_TNE;
				//cout<<"established TNE node of level "<<depth<<endl; 
				
				OctreeNode *node_TSE = new OctreeNode; 
				node_TSE->X = current->X + res_list_5cm[depth+2];
				node_TSE->Y = current->Y - res_list_5cm[depth+2]; 
				node_TSE->Z = current->Z + res_list_5cm[depth+2];
				current->TSE = node_TSE;
				//cout<<"established TSE node of level "<<depth<<endl; 

				OctreeNode *node_TSW = new OctreeNode; 
				node_TSW->X = current->X - res_list_5cm[depth+2];
				node_TSW->Y = current->Y - res_list_5cm[depth+2]; 
				node_TSW->Z = current->Z + res_list_5cm[depth+2];
				current->TSW = node_TSW;
				//cout<<"established TSW node of level "<<depth<<endl;

				OctreeNode *node_TNW = new OctreeNode; 
				node_TNW->X = current->X - res_list_5cm[depth+2];
				node_TNW->Y = current->Y + res_list_5cm[depth+2]; 
				node_TNW->Z = current->Z + res_list_5cm[depth+2];
				current->TNW = node_TNW;
				//cout<<"established TNW node of level "<<depth<<endl;

				// maintain leaf nodes index. 
				if(depth == 15){
                    leaf_nodes.push(current->BNE);
                    leaf_nodes.push(current->BSE); 
                    leaf_nodes.push(current->BSW); 
                    leaf_nodes.push(current->BNW); 
                    leaf_nodes.push(current->TNE);
                    leaf_nodes.push(current->TSE); 
                    leaf_nodes.push(current->TSW); 
                    leaf_nodes.push(current->TNW);  
				}
				
			}else{
				//cout<<depth<<" next level not null"<<endl; 
				//cout<<"next level x  "<<current->BNE->X<<endl; 
			}

			if(x>current->X && y>current->Y && z<current->Z){
				current = current->BNE; 
			}else if(x>current->X && y<current->Y && z<current->Z){
				current = current->BSE; 
			}else if(x<current->X && y<current->Y && z<current->Z){
				current = current->BSW; 
			}else if(x<current->X && y>current->Y && z<current->Z){
				current = current->BNW; 
			}else if(x>current->X && y>current->Y && z>current->Z){
				current = current->TNE; 
			}else if(x>current->X && y<current->Y && z>current->Z){
				current = current->TSE; 
			}else if(x<current->X && y<current->Y && z>current->Z){
				current = current->TSW; 
			}else if(x<current->X && y>current->Y && z>current->Z){
				current = current->TNW; 
			}else{
				//std::cout<<"something else"<<endl; 
			}
		}

		// now we are at level 16. 
		// "current" pointer indicates to the exact node the voxel should be added. 
		
		if(current->Occ == 1){	// we do not need to recognize this node as unknown 
			current->Occ = -1; 
		}else if(current->Occ > -16){
			current->Occ -= 1; 
		}else{
			// no case according to algorithm
		}
    }
    
    void inOrderTraverse(OctreeNode *node){
        if(node == NULL){
            return; 
        }
        inOrderTraverse(node->TNW);  
        inOrderTraverse(node->TNE); 
        inOrderTraverse(node->TSE); 
        inOrderTraverse(node->TSW); 
        inOrderTraverse(node->BNW); 
        inOrderTraverse(node->BNE); 
        inOrderTraverse(node->BSE); 
        inOrderTraverse(node->BSW); 
        std::cout<<"("<<node->X<<", "<<node->Y<<", "<<node->Z<<")\n";
    }

    void put_nodes_in_queue(OctreeNode *node){
        if(node == NULL){
            return; 
        }
        put_nodes_in_queue(node->TNW);  
        put_nodes_in_queue(node->TNE); 
        put_nodes_in_queue(node->TSE); 
        put_nodes_in_queue(node->TSW); 
        put_nodes_in_queue(node->BNW); 
        put_nodes_in_queue(node->BNE); 
        put_nodes_in_queue(node->BSE); 
        put_nodes_in_queue(node->BSW); 
		if(node->Occ > 0){
			leaf_nodes.push(node); 
		}
        //std::cout<<"pushing: "<<node<<std::endl; 
    }

    void postOrderDelete(){
        if(root == NULL){
            return; 
        }
        stack<OctreeNode*> node_stack; 
        stack<char> id_stack; 
        OctreeNode *current = new OctreeNode; 
        current = root; 
        node_stack.push(current); 

        while(!node_stack.empty()){
            if(current->TNW == NULL && current->TNE == NULL && current->TSE == NULL && current->TSW == NULL &&
                current->BNW == NULL && current->BNE == NULL && current->BSE == NULL && current->BSW == NULL){
                    // no children 
                    node_stack.pop(); 
                    if(!node_stack.empty()){
                        // not at root
                        current = node_stack.top(); 
                        if(id_stack.top() == '0'){
                            current->TNW = NULL; 
                            id_stack.pop(); 
                        }else if(id_stack.top() == '1'){
                            current->TNE = NULL; 
                            id_stack.pop(); 
                        }else if(id_stack.top() == '2'){
                            current->TSE = NULL; 
                            id_stack.pop(); 
                        }else if(id_stack.top() == '3'){
                            current->TSW = NULL; 
                            id_stack.pop();
                        }else if(id_stack.top() == '4'){
                            current->BNW = NULL; 
                            id_stack.pop();
                        }else if(id_stack.top() == '5'){
                            current->BNE = NULL; 
                            id_stack.pop();
                        }else if(id_stack.top() == '6'){
                            current->BSE = NULL; 
                            id_stack.pop();
                        }else{
                            current->BSW = NULL; 
                            id_stack.pop();
                        }
                }else{
                    // if we've come to the root
                    //std::cout<<"deleting: "<<root->Key<<endl; 
                    root = NULL; 

                }
            }else if(current->TNW != NULL){
                current = current->TNW; 
                node_stack.push(current); 
                id_stack.push('0');
            }else if(current->TNE != NULL){
                current = current->TNE; 
                node_stack.push(current); 
                id_stack.push('1');
            }else if(current->TSE != NULL){
                current = current->TSE; 
                node_stack.push(current); 
                id_stack.push('2');
            }else if(current->TSW != NULL){
                current = current->TSW; 
                node_stack.push(current); 
                id_stack.push('3');
            }else if(current->BNW != NULL){
                current = current->BNW; 
                node_stack.push(current); 
                id_stack.push('4');
            }else if(current->BNE != NULL){
                current = current->BNE; 
                node_stack.push(current); 
                id_stack.push('5');
            }else if(current->BSE != NULL){
                current = current->BSE; 
                node_stack.push(current); 
                id_stack.push('6');
            }else if(current->BSW != NULL){
                current = current->BSW; 
                node_stack.push(current); 
                id_stack.push('7');
            }else{}
        }
    }

    OctreeNode *searchNode(OctreeNode *node, float x, float y, float z){
        if(node == NULL){
            return NULL; 
        }else if((node->X == x) && (node->Y == y) && (node->Z == z)){
            return node; 
        }else if((node->X >= x) && (node->Y >= y) && (node->Z >= z)){
            return searchNode(node->TNW, x, y, z); 
        }else if((node->X >= x) && (node->Y >= y) && (node->Z <= z)){
            return searchNode(node->TNE, x, y, z); 
        }else if((node->X >= x) && (node->Y <= y) && (node->Z >= z)){
            return searchNode(node->TSE, x, y, z); 
        }else if((node->X >= x) && (node->Y <= y) && (node->Z <= z)){
            return searchNode(node->TSW, x, y, z); 
        }else if((node->X <= x) && (node->Y >= y) && (node->Z >= z)){
            return searchNode(node->BNW, x, y, z); 
        }else if((node->X <= x) && (node->Y >= y) && (node->Z <= z)){
            return searchNode(node->BNE, x, y, z); 
        }else if((node->X <= x) && (node->Y <= y) && (node->Z >= z)){
            return searchNode(node->BSE, x, y, z); 
        }else if(node->X <= x && node->Y <= y && node->Z <= z){
            return searchNode(node->BSW, x, y, z);
        }else{
            return node;   
        }
    }
    
    OctreeNode *findMinNode(){
        if(root == NULL){
            return NULL; 
        }
        //OctreeNode *current = new OctreeNode; 
        current = NULL; 
        current = root; 
        while(current->TNW != NULL || current->TNE != NULL || current->TSE != NULL || current->TSW != NULL 
            || current->BNW != NULL || current->BNE != NULL || current->BSE != NULL || current->BSW != NULL){
                // while the current node has any children
                if(current->TNW != NULL){
                    current = current->TNW;                        
                }else if(current->TNE != NULL){
                    current = current->TNE;                       
                }else if(current->TSE != NULL){
                    current = current->TSE;
                }else if(current->TSW != NULL){
                    current = current->TSW;          
                }else if(current->BNW != NULL){
                    current = current->BNW; 
                }else if(current->BNE != NULL){
                    current = current->BNE;                    
                }else if(current->BSE != NULL){
                    current = current->BSE;         
                }else if(current->BSW != NULL){
                    current = current->BSW;                      
                }else{
                    // No children. no case
                }
            }
        return current; 
    }
    
    OctreeNode *findMaxNode(){
        if(root == NULL){
            return NULL; 
        }else{
            OctreeNode *current = new OctreeNode; 
            current = root; 
            while(current->TNW != NULL || current->TNE != NULL || current->TSE != NULL || current->TSW != NULL 
                || current->BNW != NULL || current->BNE != NULL || current->BSE != NULL || current->BSW != NULL){
                // while the current node has any children
                if(current->BSW != NULL){
                    current = current->BSW;                        
                }else if(current->BSE != NULL){
                    current = current->BSE;                       
                }else if(current->BNE != NULL){
                    current = current->BNE;
                }else if(current->BNW != NULL){
                    current = current->BNW;          
                }else if(current->TSW != NULL){
                    current = current->TSW; 
                }else if(current->TSE != NULL){
                    current = current->TSE;                    
                }else if(current->TNE != NULL){
                    current = current->TNE;         
                }else if(current->TNW != NULL){
                    current = current->TNW;                      
                }else{
                    // No children. no case
                }
            }
            return current; 
        }
    }
        void destroy(OctreeNode *root){
            if(root == NULL){
                return;
            }else{
                destroy(root->TNW);
                destroy(root->TNE);
                destroy(root->TSE); 
                destroy(root->TSW); 
                destroy(root->BNW);
                destroy(root->BNE);
                destroy(root->BSE); 
                destroy(root->BSW); 
                delete root;
            }
        }  

    public:
        Octree(OctreeNode *node){
            root = node; 
        }

        Octree(float leaf_resolution){
            OctreeNode *root_node = new OctreeNode; 
            root_node->X = 0.00f; 
            root_node->Y = 0.00f;
            root_node->Z = 0.00f; 
            //root_node->Occ = 0;     // unknown - so zero
            //root_node->level = 0; 
            //root_node->R = 255; 
            //root_node->G = 255; 
            //root_node->B = 255; 
            //root_node->TNW = NULL;
            //root_node->TNE = NULL; 
            //root_node->TSE = NULL;
            //root_node->TSW = NULL;
            //root_node->BNW = NULL;
            //root_node->BNE = NULL; 
            //root_node->BSE = NULL;
            //oot_node->BSW = NULL;
            
            root = root_node; 
        }
        ~Octree(){
            //postOrderDelete(); 
            destroy(root); 
            std::cout<<"destroyed"<<endl; 
        }
        void insert(float x, float y, float z, uint8_t r, uint8_t g, uint8_t b){
            insertNode(x, y, z, r, g, b); 
        }

        void insert_free(float x, float y, float z){
            insert_free_Node(x, y, z); 
        }
        void display(){
            inOrderTraverse(root); 
            std::cout<<std::endl; 
        }

        void put_in_queue(){
            put_nodes_in_queue(root); 
        }

        void searchNode(float x, float y, float z){
            root = searchNode(root, x, y, z); 
        }
        OctreeNode *begin(){
            return findMinNode(); 
        }
        OctreeNode *end(){
            return findMaxNode(); 
        }
        
        OctreeNode *getRoot(){
            return root; 
        }

        OctreeNode &getRoot_reference(){
            return *root; // return root as a memory reference. 
        }     
};

const float epsilon = 0.0125; // epsilon to compare two floats. this value depends on the resolution we consider. (resolution /4 or lower )

// ==============================================================================================================================

__device__ const float resolution = 0.05f; 	// Resolution of 5 cm
__device__ const float half_resolution = resolution/2; // the half of the resolution. this value is used in calculations 
 
const float max_sensor_radius = 3.00f; 	// scan data further than this modulus value will not be considered. 
const float max_free_radius = 15.00f;   // scan data further than this will not be considered for freespace calculation 

void generate_node_arrays(int size, float* x, float* y, float* z, 
    float* x_result, float* y_result, float* z_result, float* x_round, float* y_round, float* z_round,
	float x_trans, float y_trans, float z_trans, double sin_a, double sin_b, double sin_g, double cos_a, double cos_b, double cos_g){
	
	int steps = max_sensor_radius/resolution; 
	for(int i=0; i<size; i++){ 
        if( (fabs(x[i]) < max_free_radius) and (fabs(y[i]) < max_free_radius) and (fabs(z[i]) < max_free_radius)){
                    
            // mark free space 
            float x1 = 0.025f, y1 = 0.025f, z1 = 0.025f; //this is the starting point of all 
            float x2 = x[i], y2 = y[i], z2 = z[i]; 

            float dx = abs(x2 - x1);
            float dy = abs(y2 - y1); 
            float dz = abs(z2 - z1);
            
            float xs = -1*resolution;
            float ys = -1*resolution;
            float zs = -1*resolution;
            
            if (x2 > x1) { xs = resolution; }    
            if (y2 > y1) { ys = resolution; } 
            if (z2 > z1) { zs = resolution; }

            if (dx >= dy and dx >= dz){
                // X is the driving axis
                //std::cout<<"X is driving axis"; 
                
                float py = 2 * dy - dx; 
                float pz = 2 * dz - dx;
                int sub_index = 0; 
                while (abs(x1-x2)>resolution/2 && (sub_index < steps)){
                    
                    x1 += xs; 
                    if (py >= 0){ 
                        y1 += ys; 
                        py -= 2 * dx; 
                    }
                    if (pz >= 0){
                        z1 += zs; 
                        pz -= 2 * dx; 
                    }
                    py += 2 * dy; 
                    pz += 2 * dz; 
                    
                    float x_free_temp = x1*cos_a*cos_b + y1*cos_a*sin_b*sin_g - y1*sin_a*cos_g + z1*cos_a*sin_b*cos_g + z1*sin_a*sin_g + x_trans;  
                    float y_free_temp = x1*sin_a*cos_b + y1*sin_a*sin_b*sin_g + y1*cos_a*cos_g + z1*sin_a*sin_b*cos_g - z1*cos_a*sin_g + y_trans; 
                    float z_free_temp = x1*sin_b*-1 + y1*cos_b*sin_g + z1*cos_b*cos_g + z_trans; 
                    
                    x_result[i*steps + sub_index] = (ceil(x_free_temp / resolution))*0.05 - half_resolution;
                    y_result[i*steps + sub_index] = (ceil(y_free_temp / resolution))*0.05 - half_resolution;  
                    z_result[i*steps + sub_index] = (ceil(z_free_temp / resolution))*0.05 - half_resolution; 
                    sub_index +=1; 
                    
                }
                
                for(int j=sub_index; j<steps; j++){
                    x_result[i*steps + j] = 0.00f; 
                    y_result[i*steps + j] = 0.00f; 
                    z_result[i*steps + j] = 0.00f; 
                }
            }
            else if(dy >= dx and dy >= dz){
                // Y axis is the driving axis
                float px = 2 * dx - dy; 
                float pz = 2 * dz - dy; 
                int sub_index = 0;
                while (abs(y1-y2)>resolution/2 && (sub_index < steps)){ 
                    //std::cout<<x1<<"\t"<<y1<<"\t"<<z1<<std::endl;
                    y1 += ys; 
                    if (px >= 0){ 
                        x1 += xs; 
                        px -= 2 * dy;
                    }
                    if (pz >= 0){ 
                        z1 += zs; 
                        pz -= 2 * dy;
                    } 
                    px += 2 * dx; 
                    pz += 2 * dz;

                    float x_free_temp = x1*cos_a*cos_b + y1*cos_a*sin_b*sin_g - y1*sin_a*cos_g + z1*cos_a*sin_b*cos_g + z1*sin_a*sin_g + x_trans;  
                    float y_free_temp = x1*sin_a*cos_b + y1*sin_a*sin_b*sin_g + y1*cos_a*cos_g + z1*sin_a*sin_b*cos_g - z1*cos_a*sin_g + y_trans; 
                    float z_free_temp = x1*sin_b*-1 + y1*cos_b*sin_g + z1*cos_b*cos_g + z_trans; 
                    
                    x_result[i*steps + sub_index] = (ceil(x_free_temp / resolution))*0.05 - half_resolution;
                    y_result[i*steps + sub_index] = (ceil(y_free_temp / resolution))*0.05 - half_resolution;  
                    z_result[i*steps + sub_index] = (ceil(z_free_temp / resolution))*0.05 - half_resolution; 
                    sub_index +=1; 
                }
                
                for(int j=sub_index; j<steps; j++){
                    x_result[i*steps + j] = 0.00f; 
                    y_result[i*steps + j] = 0.00f; 
                    z_result[i*steps + j] = 0.00f; 
                } 
            }
            else{
                // Z axis is the driving axis
                
                float py = 2*dy - dz;       // slope error 
                float px = 2*dx - dz; 
                int sub_index = 0;
                while(abs(z1-z2)>resolution/2 && (sub_index < steps)){
                    //std::cout<<x1<<"\t"<<y1<<"\t"<<z1<<std::endl;
                    z1 += zs; 
                    if (py >= 0){ 
                        y1 += ys; 
                        py -= 2*dz; 
                    }
                    if (px >= 0){ 
                        x1 += xs ;
                        px -= 2*dz; 
                    }
                    py += 2*dy; 
                    px += 2*dx; 
                    
                    float x_free_temp = x1*cos_a*cos_b + y1*cos_a*sin_b*sin_g - y1*sin_a*cos_g + z1*cos_a*sin_b*cos_g + z1*sin_a*sin_g + x_trans;  
                    float y_free_temp = x1*sin_a*cos_b + y1*sin_a*sin_b*sin_g + y1*cos_a*cos_g + z1*sin_a*sin_b*cos_g - z1*cos_a*sin_g + y_trans; 
                    float z_free_temp = x1*sin_b*-1 + y1*cos_b*sin_g + z1*cos_b*cos_g + z_trans; 
                    
                    x_result[i*steps + sub_index] = (ceil(x_free_temp / resolution))*0.05 - half_resolution;
                    y_result[i*steps + sub_index] = (ceil(y_free_temp / resolution))*0.05 - half_resolution;  
                    z_result[i*steps + sub_index] = (ceil(z_free_temp / resolution))*0.05 - half_resolution; 
                    
                    sub_index +=1; 
                }
                
                for(int j=sub_index; j<steps; j++){
                    x_result[i*steps + j] = 0.00f; 
                    y_result[i*steps + j] = 0.00f; 
                    z_result[i*steps + j] = 0.00f; 
                } 
            }	
            
            // generate the obstacles array. 
            if( (fabs(x[i]) < max_sensor_radius) and (fabs(y[i]) < max_sensor_radius) and (fabs(z[i]) < max_sensor_radius)){
                float x_temp = x[i]*cos_a*cos_b + y[i]*cos_a*sin_b*sin_g - y[i]*sin_a*cos_g + z[i]*cos_a*sin_b*cos_g + z[i]*sin_a*sin_g + x_trans;  
                float y_temp = x[i]*sin_a*cos_b + y[i]*sin_a*sin_b*sin_g + y[i]*cos_a*cos_g + z[i]*sin_a*sin_b*cos_g - z[i]*cos_a*sin_g + y_trans; 
                float z_temp = x[i]*sin_b*-1 + y[i]*cos_b*sin_g + z[i]*cos_b*cos_g + z_trans; 
                
                x_round[i] = (ceilf(x_temp / resolution))*0.05 - half_resolution;
                y_round[i] = (ceilf(y_temp / resolution))*0.05 - half_resolution; 
                z_round[i] = (ceilf(z_temp / resolution))*0.05 - half_resolution;
            }else{
                x_round[i] = 0.00f;
                y_round[i] = 0.00f; 
                z_round[i] = 0.00f; 
            }
        }else{
            for(int j=0; j<steps; j++){
                x_result[i*steps +j] = 0.00f; 
                y_result[i*steps +j] = 0.00f; 
                z_result[i*steps +j] = 0.00f; 
            }
            x_round[i] = 0.00f;
            y_round[i] = 0.00f; 
            z_round[i] = 0.00f;   
        } 	
    }
}

Octree tree(half_resolution);  

int cudamain(sensor_msgs::PointCloud2 point_cloud_std, nav_msgs::Odometry odom_message_std, int size){ 
    double starttotal, endtotal; 
	starttotal = clock();
    // end4 = clock();
    // double time4 = (double)(end4 - start4);
    double start1, end1; 
	start1 = clock();
    // end4 = clock();
	// double time4 = (double)(end4 - start4);
	//make_range_array(resolution, max_sensor_radius); 
	int array_size = size; 	 

	// convert quaternion orientation into roll, pitch, yaw representation 
	//double roll, pitch, yaw;
	double roll, pitch, yaw; 
	tf::Quaternion quat;
    tf::quaternionMsgToTF(odom_message_std.pose.pose.orientation, quat);	
    tf::Matrix3x3(quat).getRPY(roll, pitch, yaw);
	float x_position = (float) odom_message_std.pose.pose.position.x; 
	float y_position = (float) odom_message_std.pose.pose.position.y; 
	float z_position = (float) odom_message_std.pose.pose.position.z; 

	double sin_gamma = sin(roll); 
	double sin_beta = sin(pitch); 
	double sin_alpha = sin(yaw); 
	double cos_gamma = cos(roll); 
	double cos_beta = cos(pitch); 
	double cos_alpha = cos(yaw); 

	
	//std::cout<<"alpha: "<<yaw<<"  sin alpha: "<<sin_alpha<<std::endl;
	//std::cout<<"\n\n"; 

	int counter = 0; 
	int effective_point_count = 0; 
	//declare the arrray sets before reading the point cloud values 
	
	float *x, *y, *z; // for allocating position values of the points 
	float *x_rounded, *y_rounded, *z_rounded; 							// the intermediate results after rounding off the x, y, z, original values to the resolution 
	u_int8_t *r, *g, *b; // for color values of the point cloud 
	float *x_free, *y_free, *z_free;

	int size_position = array_size * sizeof(float);
	int size_color = array_size * sizeof(u_int8_t);
    int max_step_count = max_sensor_radius/resolution; 
    int max_free_voxel_count = array_size * max_sensor_radius/resolution; 
	int size_free_array = max_free_voxel_count * sizeof(float); 

	x = (float *)malloc( size_position );
   	y = (float *)malloc( size_position );
	z = (float *)malloc( size_position );

	x_rounded = (float *)malloc( size_position );
   	y_rounded = (float *)malloc( size_position );
	z_rounded = (float *)malloc( size_position );

	r = (u_int8_t *)malloc( size_color );
    g = (u_int8_t *)malloc( size_color );
	b = (u_int8_t *)malloc( size_color );
    
    x_free = (float *)malloc( size_free_array );
	y_free = (float *)malloc( size_free_array );
	z_free = (float *)malloc( size_free_array );
    
    // positional data vector generation 
	for(sensor_msgs::PointCloud2ConstIterator<float> it(point_cloud_std, "x"); it!=it.end(); ++it){
		y[counter] = it[0] * -1; 
		z[counter] = it[1] * -1;
		x[counter] = it[2];
		counter+=1;  
		
	}
	counter = 0; 
    for(sensor_msgs::PointCloud2ConstIterator<u_int8_t> it_color(point_cloud_std, "rgb"); it_color!=it_color.end(); ++it_color){
		b[counter] = unsigned(it_color[0]);	
		g[counter] = unsigned(it_color[1]);	
		r[counter] = unsigned(it_color[2]); 
		counter+=1; 
	}
	counter = 0; 
	
	// double start1, end1; 
	// start1 = clock();
    end1 = clock();
	double time1 = (double)(end1 - start1);
	
    double start2, end2; 
	start2 = clock();
    // end1 = clock();
	// double time1 = (double)(end1 - start1);
    generate_node_arrays(array_size, x, y, z, x_free, y_free, z_free, x_rounded, y_rounded, z_rounded, 
        x_position, y_position, z_position, sin_alpha, sin_beta, sin_gamma, cos_alpha, cos_beta, cos_gamma);
    
    // double start1, end1; 
	// start1 = clock();
    end2 = clock();
	double time2 = (double)(end2 - start2);
        
    current = new OctreeNode;   //initiating OctreeNodes
    parent = new OctreeNode; 
    //Octree tree(half_resolution);               // create the new tree for every scan instance 
    
    double start3, end3; 
	start3 = clock();
    // end1 = clock();
	// double time1 = (double)(end1 - start1);

    for(int i=0; i<array_size; i++){
        if(x_rounded[i] != 0.00f){
            tree.insert(x_rounded[i], y_rounded[i], z_rounded[i], r[i], g[i], b[i]); 
        }
    }
    // double start1, end1; 
	// start1 = clock();
    end3 = clock();
	double time3 = (double)(end3 - start3);
    
    double start4, end4; 
	start4 = clock();
    // end1 = clock();
	// double time1 = (double)(end1 - start1);

    for(int i=0; i<max_free_voxel_count; i+=max_step_count){
        for(int j=0; j<max_step_count-1; j++){
            int k = i+j; 
            if(x_free[k+1] == 0.00f){
                //std::cout<<"last voxel: "<<k+1<<endl; 
                break; 
            }else{
                //if(j ==59){std::cout<<"at the edge"<<std::endl; }
                tree.insert_free(x_free[k], y_free[k], z_free[k]);  
            }
        }
    }		
    
	// double start3, end3; 
	// start4 = clock();
    end4 = clock();
	double time4 = (double)(end4 - start4);
    
    double start5, end5; 
	start5 = clock();
    // end1 = clock();
	// double time1 = (double)(end1 - start1);

    
    std::queue<OctreeNode*> temp_nodes; 
	temp_nodes = leaf_nodes; 
    
    std::string map_file_name = "octree_nodes.txt";
	ofstream offile;
	offile.open(map_file_name.c_str(), ios::trunc);
	if(offile.is_open()) { 
		while(!temp_nodes.empty()){
            current = temp_nodes.front(); 
            if(current->Occ >0){
				offile<<current->X<<"\t"<<current->Y<<"\t"<<current->Z<<"\t"<<unsigned(current->R)<<"\t"<<unsigned(current->G)<<"\t"<<unsigned(current->B)<<std::endl;  
			}
            temp_nodes.pop(); 
        }				
	}
	
	offile.close();
    
    // double start3, end3; 
	// start3 = clock();
    end5 = clock();
	double time5 = (double)(end5 - start5);
    
    // double start3, end3; 
	// start3 = clock();
    endtotal = clock();
	double timetotal = (double)(endtotal - starttotal);

    std::cout<<time1<<"\t"<<time2<<"\t"<<time3<<"\t"<<time4<<"\t"<<time5<<"\t"<<timetotal<<endl; 
    
    free(x);
    free(y);
	free(z);
	free(r);
	free(g);
	free(b);
	free(x_rounded); 
	free(y_rounded); 
    free(z_rounded);
    
    free(x_free); 
    free(y_free); 
    free(z_free); 

	return EXIT_SUCCESS; 	
}
