#include "hip/hip_runtime.h"
// This is the real Hello World for CUDA!
//It takes the string 'Hello', prints it, then passes it to cuda with an array of offsets
// Then the offsets are added in parallel to produce the string world! 
#include <iostream>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ros/ros.h"
#include "sensor_msgs/PointCloud2.h"
#include <sensor_msgs/PointCloud.h>
#include <sensor_msgs/point_cloud_conversion.h>
#include <time.h>
#include <sensor_msgs/point_cloud2_iterator.h>
#include <string>
#include <cmath> 
#include "math.h"
#include <cstdlib>
#include <stdio.h>
#include <map>
#include <iterator>
#include <boost/lexical_cast.hpp>
#include <tf/transform_datatypes.h>
#include <nav_msgs/Odometry.h>
#include <geometry_msgs/Vector3.h>
#include "geometry_msgs/Quaternion.h"

// =========================================================================================================================
// This class file is to transport into include directory after development 


class Key3D{
	
private: 
	float x; 
	float y; 
	float z; 

public: 
	//Null constructor
	__host__ __device__ Key3D(){} 

	//preferred constructor
	__host__ __device__ Key3D(float x_in, float y_in, float z_in){
		x = x_in; 
		y = y_in; 
		z = z_in;  
	}

	//retrieve values for testing
	__host__ __device__ const float& getX() const {
		return x; 
	}
	__host__ __device__ const float& getY() const {
		return y; 
	}
	__host__ __device__ const float& getZ() const {
		return z; 
	}

	//the sorting operator for a map
	bool operator< (const Key3D& keyObj) const{

        if(keyObj.x < this->x){
			return true;
		} 
		return false;         
    }

	//destructor
	__host__ __device__ ~Key3D(){} 

}; 

const float epsilon = 0.0125; // epsilon to compare two floats. this value depends on the resolution we consider. (resolution /4 or lower )

struct VoxelKeyComparator
{	
	bool operator()(const Key3D &left, const Key3D &right) const{
		
		if((abs(left.getX()-right.getX()) > epsilon) || (abs(left.getY()-right.getY()) > epsilon) || (abs(left.getZ()-right.getZ()) > epsilon)){
			return (left.getX() <= right.getX());
			//return true; 
		}
		return false; 
	}
};

// This class is originally developed to generate within host. updating count parameter with device might require extra effort. 
class VoxelBody{		
private: 
	int point_count; 
	uint8_t r; 
	uint8_t g; 
	uint8_t b; 

public: 
	//Null constructor

	__host__ __device__ VoxelBody(){
		point_count = 0; 
		r = 0; 
		g = 0; 
		b = 0; 
	} 
	// means a negative voxel
	__host__ __device__ VoxelBody(bool free){
		
		point_count = -1; 
		r = 0; 
		g = 255; 
		b = 0; 
	}

	//full constructor
	__host__ __device__ VoxelBody(uint8_t r_in, uint8_t g_in, uint8_t b_in){
		point_count = 1; 	
		r = r_in; 
		g = g_in; 
		b = b_in; 						
	}
	// desctructor
	__host__ __device__ ~VoxelBody(){} 

	__host__ __device__ void updateVoxel(uint8_t r_in, uint8_t g_in, uint8_t b_in){
		
		if(point_count < 10){
			point_count +=1; 
		}

		r = (r + r_in)/2; 
		g = (g + g_in)/2; 
		b = (b + b_in)/2;
	}
	
	__host__ __device__ void updateVoxel(bool isoccupied){
		
		if(point_count > -10){
			point_count -=1; 
		}
	}


	__host__ __device__ const uint8_t& get_r() const { return r; }
	__host__ __device__ const uint8_t& get_g() const { return g; }
	__host__ __device__ const uint8_t& get_b() const { return b; }
	__host__ __device__ const int& get_point_count() const {return point_count; }
	
};

// ==============================================================================================================================

using namespace std; 

#define THREADS_PER_BLOCK 256		// the optimal value is number of cuda cores, if (#cuda cores < max th.pr.blck). 256 for TX2

const float resolution = 0.05f; 	// Resolution of 5 cm
const float half_resolution = resolution/2; // the half of the resolution. this value is used in calculations 

const float max_sensor_radius = 3.00f; 	// scan data further than this modulus value will not be considered. 
 

void serial_round_off_positional_coords(int size, float* x, float* y, float* z, float* x_result, float* y_result, float* z_result, 
	float x_trans, float y_trans, float z_trans, double sin_a, double sin_b, double sin_g, double cos_a, double cos_b, double cos_g, 
	float *x_freespace, float *y_freespace, float *z_freespace){
	
	int steps = max_sensor_radius/resolution;	// how many max steps for free space tracking 
	
	for(int i=0; i<size; i++){
		if( (abs(x[i]) < max_sensor_radius) and (abs(y[i]) < max_sensor_radius) and (abs(z[i]) < max_sensor_radius)){
			 		
			// mark free space 
			float x1 = 0.025f, y1 = 0.025f, z1 = 0.025f; //this is the starting point of all 
			float x2 = x[i], y2 = y[i], z2 = z[i]; 

			float dx = abs(x2 - x1);
			float dy = abs(y2 - y1); 
			float dz = abs(z2 - z1);
			
			float xs = -1*resolution;
			float ys = -1*resolution;
			float zs = -1*resolution;
			
			if (x2 > x1) { xs = resolution; }    
			if (y2 > y1) { ys = resolution; } 
			if (z2 > z1) { zs = resolution; }

			if (dx >= dy and dx >= dz){
				// X is the driving axis
				//std::cout<<"X is driving axis"; 
				
				float py = 2 * dy - dx; 
				float pz = 2 * dz - dx;
				int sub_index = 0; 
				while (abs(x1-x2)>resolution/2){
					
					x1 += xs; 
					if (py >= 0){ 
						y1 += ys; 
						py -= 2 * dx; 
					}
					if (pz >= 0){
						z1 += zs; 
						pz -= 2 * dx; 
					}
					py += 2 * dy; 
					pz += 2 * dz; 
					
					float x_free_temp = x1*cos_a*cos_b + y1*cos_a*sin_b*sin_g - y1*sin_a*cos_g + z1*cos_a*sin_b*cos_g + z1*sin_a*sin_g + x_trans;  
					float y_free_temp = x1*sin_a*cos_b + y1*sin_a*sin_b*sin_g + y1*cos_a*cos_g + z1*sin_a*sin_b*cos_g - z1*cos_a*sin_g + y_trans; 
					float z_free_temp = x1*sin_b*-1 + y1*cos_b*sin_g + z1*cos_b*cos_g + z_trans; 
					
					x_freespace[i*steps + sub_index] = (ceil(x_free_temp / resolution))*0.05 - half_resolution;
					y_freespace[i*steps + sub_index] = (ceil(y_free_temp / resolution))*0.05 - half_resolution;  
					z_freespace[i*steps + sub_index] = (ceil(z_free_temp / resolution))*0.05 - half_resolution; 
					sub_index +=1; 
				}
				 
				for(int j=sub_index; j<steps; j++){
					x_freespace[i*steps + j] = 0.00f; 
					y_freespace[i*steps + j] = 0.00f; 
					z_freespace[i*steps + j] = 0.00f; 
				}
			}
			else if(dy >= dx and dy >= dz){
				// Y axis is the driving axis
				std::cout<<"y is driving axis";
				
				float px = 2 * dx - dy; 
				float pz = 2 * dz - dy; 
				int sub_index = 0;
				while (abs(y1-y2)>resolution/2){ 
					//std::cout<<x1<<"\t"<<y1<<"\t"<<z1<<std::endl;
					y1 += ys; 
					if (px >= 0){ 
						x1 += xs; 
						px -= 2 * dy;
					}
					if (pz >= 0){ 
						z1 += zs; 
						pz -= 2 * dy;
					} 
					px += 2 * dx; 
					pz += 2 * dz;

					float x_free_temp = x1*cos_a*cos_b + y1*cos_a*sin_b*sin_g - y1*sin_a*cos_g + z1*cos_a*sin_b*cos_g + z1*sin_a*sin_g + x_trans;  
					float y_free_temp = x1*sin_a*cos_b + y1*sin_a*sin_b*sin_g + y1*cos_a*cos_g + z1*sin_a*sin_b*cos_g - z1*cos_a*sin_g + y_trans; 
					float z_free_temp = x1*sin_b*-1 + y1*cos_b*sin_g + z1*cos_b*cos_g + z_trans; 
					
					x_freespace[i*steps + sub_index] = (ceil(x_free_temp / resolution))*0.05 - half_resolution;
					y_freespace[i*steps + sub_index] = (ceil(y_free_temp / resolution))*0.05 - half_resolution;  
					z_freespace[i*steps + sub_index] = (ceil(z_free_temp / resolution))*0.05 - half_resolution; 
					sub_index +=1; 
				}
				
				for(int j=sub_index; j<steps; j++){
					x_freespace[i*steps + j] = 0.00f; 
					y_freespace[i*steps + j] = 0.00f; 
					z_freespace[i*steps + j] = 0.00f; 
				} 
			}
			else{
				// Z axis is the drivinf axis
				std::cout<<"z is driving axis";
				
				float py = 2*dy - dz;       // slope error 
				float px = 2*dx - dz; 
				int sub_index = 0;
				while(abs(z1-z2)>resolution/2){
					//std::cout<<x1<<"\t"<<y1<<"\t"<<z1<<std::endl;
					z1 += zs; 
					if (py >= 0){ 
						y1 += ys; 
						py -= 2*dz; 
					}
					if (px >= 0){ 
						x1 += xs ;
						px -= 2*dz; 
					}
					py += 2*dy; 
					px += 2*dx; 
					
					float x_free_temp = x1*cos_a*cos_b + y1*cos_a*sin_b*sin_g - y1*sin_a*cos_g + z1*cos_a*sin_b*cos_g + z1*sin_a*sin_g + x_trans;  
					float y_free_temp = x1*sin_a*cos_b + y1*sin_a*sin_b*sin_g + y1*cos_a*cos_g + z1*sin_a*sin_b*cos_g - z1*cos_a*sin_g + y_trans; 
					float z_free_temp = x1*sin_b*-1 + y1*cos_b*sin_g + z1*cos_b*cos_g + z_trans; 
					
					x_freespace[i*steps + sub_index] = (ceil(x_free_temp / resolution))*0.05 - half_resolution;
					y_freespace[i*steps + sub_index] = (ceil(y_free_temp / resolution))*0.05 - half_resolution;  
					z_freespace[i*steps + sub_index] = (ceil(z_free_temp / resolution))*0.05 - half_resolution; 
					
					sub_index +=1; 
				}
				 
				for(int j=sub_index; j<steps; j++){
					x_freespace[i*steps + j] = 0.00f; 
					y_freespace[i*steps + j] = 0.00f; 
					z_freespace[i*steps + j] = 0.00f; 
				} 
			}

			
			//B: Transformation model 1 -----------yaw only----------- for flat surface navigation-----------------------------------
			/*float x_temp = x[i]*cos_a - y[i]*sin_a + x_trans;  
			float y_temp = x[i]*sin_a + y[i]*cos_a + y_trans; 
			float z_temp = z[i] + z_trans; 
			
			x_result[i] = (ceilf(x_temp / resolution))*0.05 - half_resolution;
			y_result[i] = (ceilf(y_temp / resolution))*0.05 - half_resolution; 
			z_result[i] = (ceilf(z_temp / resolution))*0.05 - half_resolution; */
			//------------------------------------------------------------------------------------------------
			
			// C: Transformation model 2 ---------- Roll, pitch, yaw combined--------- for inclined planes navigation --------------
			float x_temp = x[i]*cos_a*cos_b + y[i]*cos_a*sin_b*sin_g - y[i]*sin_a*cos_g + z[i]*cos_a*sin_b*cos_g + z[i]*sin_a*sin_g + x_trans;  
			float y_temp = x[i]*sin_a*cos_b + y[i]*sin_a*sin_b*sin_g + y[i]*cos_a*cos_g + z[i]*sin_a*sin_b*cos_g - z[i]*cos_a*sin_g + y_trans; 
			float z_temp = x[i]*sin_b*-1 + y[i]*cos_b*sin_g + z[i]*cos_b*cos_g + z_trans; 
			
			x_result[i] = (ceil(x_temp / resolution))*0.05 - half_resolution;
			y_result[i] = (ceil(y_temp / resolution))*0.05 - half_resolution; 
			z_result[i] = (ceil(z_temp / resolution))*0.05 - half_resolution;
			// -----------------------------------------------------------------------------------------------------------------------------------------
			
		}else{
			x_result[i] = 0.00f; 
			y_result[i] = 0.00f; 
			z_result[i] = 0.00f; 

		} 
	}
	
}


void serial_remove_null_points(int camera_array_size, float* x, float* y, float* z, uint8_t* r, uint8_t* g, uint8_t* b, 
													float* x_cropped, float* y_cropped, float* z_cropped, uint8_t* r_cropped, uint8_t* g_cropped, uint8_t* b_cropped){
	// x, y, z are input arrays with rounded and range controlled poitn clouds
	// x_cropped etc. stores the new array with 0,0,0 points removed. 
	int index = 0; 
	for(int i=0; i<camera_array_size; i++){			// size of original set of point cloud
		if(x[i] != 0.00f){
			x_cropped[index] = x[i]; 
			y_cropped[index] = y[i]; 
			z_cropped[index] = z[i]; 
			r_cropped[index] = r[i]; 
			g_cropped[index] = g[i]; 
			b_cropped[index] = b[i]; 
			index +=1; 
		}
	}
}

static std::map<Key3D, VoxelBody, VoxelKeyComparator> voxel_map_downsampled, freespace_map; 

void serial_downsample_point_cloud(int array_size, float* x_cleaned, float* y_cleaned, float* z_cleaned, uint8_t* r_cleaned, uint8_t* g_cleaned, uint8_t* b_cleaned){

	Key3D temp_key; 
	VoxelBody temp_body; 
	
	for(int i=0; i<array_size; i++){
		temp_key = Key3D(x_cleaned[i], y_cleaned[i], z_cleaned[i]);
		temp_body = VoxelBody(r_cleaned[i], g_cleaned[i], b_cleaned[i]); 
		// Keep inerting for now. will genaralize rgb values and count later. 
		voxel_map_downsampled.insert(std::make_pair<Key3D, VoxelBody>(temp_key, temp_body));
	}
	
}
void serial_add_freespace(int array_size, float *x_in, float *y_in, float *z_in){
	for(int i=0; i<array_size; i++){
		Key3D temp_key; 
		VoxelBody temp_body; 

		if(x_in[i] != 0.00f){
			// effective points 
			Key3D temp_key = Key3D(x_in[i], y_in[i], z_in[i]); 
			VoxelBody temp_body = VoxelBody(false); 
			std::map<Key3D, VoxelBody, VoxelKeyComparator>::iterator it; 
			it = freespace_map.find(temp_key); 
			if(it !=freespace_map.end()){
				it->second.updateVoxel(false); 
			}else{
				freespace_map.insert(std::make_pair<Key3D, VoxelBody>(temp_key, temp_body));
			}

		}
	}
}

int downsampled_count = 0; 
void readings_function(int array_size, float* x_cleaned, float* y_cleaned, float* z_cleaned, uint8_t* r_cleaned, uint8_t* g_cleaned, uint8_t* b_cleaned, 
							std::map<Key3D, VoxelBody, VoxelKeyComparator> map_reading){

	Key3D temp_key; 
	VoxelBody temp_body; 
	
	for(int i=0; i<array_size; i++){
		temp_key = Key3D(x_cleaned[i], y_cleaned[i], z_cleaned[i]);
		temp_body = VoxelBody(r_cleaned[i], g_cleaned[i], b_cleaned[i]); 
		// Keep inerting for now. will genaralize rgb values and count later. 
		map_reading.insert(std::make_pair<Key3D, VoxelBody>(temp_key, temp_body));
	}
	downsampled_count =0; 
	
	std::map<Key3D, VoxelBody, VoxelKeyComparator>::iterator itr; 
	for(itr=map_reading.begin(); itr!=map_reading.end(); itr++){
		downsampled_count +=1;   
	}

}


int cudamain(sensor_msgs::PointCloud2 point_cloud_std, nav_msgs::Odometry odom_message_std, int size){ 
	//make_range_array(resolution, max_sensor_radius); 
	int array_size = size; 	 

	// convert quaternion orientation into roll, pith, yaw representation 
	double start_prep, end_prep; 
	start_prep = clock(); //----------------------------------------------------------------------
	double roll, pitch, yaw; 
	tf::Quaternion quat;
    tf::quaternionMsgToTF(odom_message_std.pose.pose.orientation, quat);	
    tf::Matrix3x3(quat).getRPY(roll, pitch, yaw);
	float x_position = (float) odom_message_std.pose.pose.position.x; 
	float y_position = (float) odom_message_std.pose.pose.position.y; 
	float z_position = (float) odom_message_std.pose.pose.position.z; 

	double sin_gamma = sin(roll); 
	double sin_beta = sin(pitch); 
	double sin_alpha = sin(yaw); 
	double cos_gamma = cos(roll); 
	double cos_beta = cos(pitch); 
	double cos_alpha = cos(yaw); 

	
	//std::cout<<"alpha: "<<yaw<<"  sin alpha: "<<sin_alpha<<std::endl;
	//std::cout<<"\n\n"; 

	int counter = 0; 
	int effective_point_count = 0; 
	//declare the arrray sets before reading the point cloud values 
	
	float *x, *y, *z; // for allocating position values of the points 
	float *x_rounded, *y_rounded, *z_rounded; 							// the intermediate results after rounding off the x, y, z, original values to the resolution 
	u_int8_t *r, *g, *b; // for color values of the point cloud 
	float *x_free, *y_free, *z_free; 
	
	int step_count =  max_sensor_radius/resolution; 
	int size_position = array_size * sizeof(float);
	int size_color = array_size * sizeof(u_int8_t);
	
	x = (float *)malloc( size_position );
   	y = (float *)malloc( size_position );
	z = (float *)malloc( size_position );

	x_rounded = (float *)malloc( size_position );
   	y_rounded = (float *)malloc( size_position );
	z_rounded = (float *)malloc( size_position );
	
	x_free = (float *)malloc( size_position*step_count);
	y_free = (float *)malloc( size_position*step_count);
	z_free = (float *)malloc( size_position*step_count);
	
	r = (u_int8_t *)malloc( size_color );
    g = (u_int8_t *)malloc( size_color );
	b = (u_int8_t *)malloc( size_color );
	
	// positional data vector generation 
	for(sensor_msgs::PointCloud2ConstIterator<float> it(point_cloud_std, "x"); it!=it.end(); ++it){
		y[counter] = it[0] * -1; 
		z[counter] = it[1] * -1;
		x[counter] = it[2];
		counter+=1;  
		
	}
	counter = 0; 
    for(sensor_msgs::PointCloud2ConstIterator<u_int8_t> it_color(point_cloud_std, "rgb"); it_color!=it_color.end(); ++it_color){
		b[counter] = unsigned(it_color[0]);	
		g[counter] = unsigned(it_color[1]);	
		r[counter] = unsigned(it_color[2]); 
		counter+=1; 
	}
	counter = 0; 
	end_prep = clock(); //--------------------------------------------------------------------------------------------------------------
	double time_prep = (double)(end_prep - start_prep); 
	
	
	double start_round, end_round; 
	start_round = clock(); //-------------------------------------------------------------------------------------------------------------
	serial_round_off_positional_coords( array_size, x, y, z, x_rounded, y_rounded, z_rounded,
									x_position, y_position, z_position, sin_alpha, sin_beta, sin_gamma, cos_alpha, cos_beta, cos_gamma, 
									x_free, y_free, z_free);
	end_round = clock();	//------------------------------------------------------------------------------------------------------------
	double time_round = (double)(end_round-start_round); 
	
	free(x); 	// finished using these arrays 
	free(y); 
	free(z); 

	/*for(int k=0; k<array_size*step_count; k++){
		std::cout<<k<<"\t"<<x_free[k]<<"\t"<<y_free[k]<<"\t"<<z_free[k]<<std::endl; 
	}*/
	
	for(int i=0; i<array_size; i++){		// counted the number of informative points 
		if(x_rounded[i] != 0.00f){
			effective_point_count += 1; 
		}
	}

	float *x_cleaned, *y_cleaned, *z_cleaned; 
	uint8_t *r_cleaned, *g_cleaned, *b_cleaned; 

	int size_cleaned = effective_point_count * sizeof(float);
	int size_colour_cleaned = effective_point_count * sizeof(uint8_t); 
	
	x_cleaned = (float *)malloc( size_cleaned );
	y_cleaned = (float *)malloc( size_cleaned );
	z_cleaned = (float *)malloc( size_cleaned );
	r_cleaned = (uint8_t *)malloc(size_colour_cleaned); 
	g_cleaned = (uint8_t *)malloc(size_colour_cleaned); 
	b_cleaned = (uint8_t *)malloc(size_colour_cleaned); 

	double start_null, end_null; 
	start_null = clock(); //--------------------------------------------------------------------------------------------------------------------------
	serial_remove_null_points(array_size, x_rounded, y_rounded, z_rounded, r, g, b, x_cleaned, y_cleaned, z_cleaned, r_cleaned, g_cleaned, b_cleaned);  
	end_null = clock();  // --------------------------------------------------------------------------------------------------------------------------
	double time_null = (double) (end_null - start_null); 

	free(x_rounded); 	// finished using these arrays 
	free(y_rounded); 
	free(z_rounded);
	free(r); 
	free(g); 
	free(b);  

	double start_downsample, end_downsample; 
	start_downsample = clock(); //------------------------------------------------------------------------------------------------------------------------
	serial_downsample_point_cloud(effective_point_count, x_cleaned, y_cleaned, z_cleaned, r_cleaned, g_cleaned, b_cleaned); 
	
	end_downsample = clock(); //--------------------------------------------------------------------------------------
	double time_downsample = (double)(end_downsample-start_downsample); 
	
	serial_add_freespace(array_size*step_count, x_free, y_free, z_free); 
	/*
	std::string output_file_name = "result_map_key_generation.txt";
	ofstream offile;
	offile.open(output_file_name.c_str(), ios::trunc);
	if(offile.is_open()) { 
		std::map<Key3D, VoxelBody, VoxelKeyComparator>::iterator itr; 
		for(itr=voxel_map_downsampled.begin(); itr!=voxel_map_downsampled.end(); itr++){
			offile<<itr->first.getX()<<"\t"<<itr->first.getY()<<"\t"<<itr->first.getZ()<<"\t"<<unsigned(itr->second.get_r())<<"\t"<<unsigned(itr->second.get_g())<<"\t"<<unsigned(itr->second.get_b())<<std::endl;  
		}
		offile<<"\n\n"; 
		//std::cout<<"data written"<<std::endl; 											
								
	}
	offile.close(); */

	
	std::string output_file_name = "freespace.txt";
	ofstream offile;
	offile.open(output_file_name.c_str(), ios::trunc);
	if(offile.is_open()) { 
		std::map<Key3D, VoxelBody, VoxelKeyComparator>::iterator itr; 
		for(itr=freespace_map.begin(); itr!=freespace_map.end(); itr++){
			offile<<itr->first.getX()<<"\t"<<itr->first.getY()<<"\t"<<itr->first.getZ()<<"\t"<<itr->second.get_point_count()<<std::endl;  
		}
		offile<<"\n\n"; 
		//std::cout<<"data written"<<std::endl; 											
								
	}
	offile.close(); 


	std::cout<<time_prep<<"\t"<<time_round<<"\t"<<time_null<<"\t"<<time_downsample<<std::endl; 
	 
	free(x_free); 
	free(y_free); 
	free(z_free); 
	free(x_cleaned); 
	free(y_cleaned); 
	free(z_cleaned); 
	free(r_cleaned); 
	free(g_cleaned); 
	free(b_cleaned); 

	return EXIT_SUCCESS; 	
}
