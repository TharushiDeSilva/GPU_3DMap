#include "hip/hip_runtime.h"
// This is the real Hello World for CUDA!
//It takes the string 'Hello', prints it, then passes it to cuda with an array of offsets
// Then the offsets are added in parallel to produce the string world! 
#include <iostream>
#include <bits/stdc++.h> 
#include <stdint.h>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "ros/ros.h"
#include "sensor_msgs/PointCloud2.h"
#include <sensor_msgs/PointCloud.h>
#include <sensor_msgs/point_cloud_conversion.h>
#include <time.h>
#include <sensor_msgs/point_cloud2_iterator.h>
#include <string>
#include <cmath> 
#include "math.h"
#include <cstdlib>
#include <stdio.h>
#include <map>
#include <iterator>
#include <boost/lexical_cast.hpp>
#include <tf/transform_datatypes.h>
#include <nav_msgs/Odometry.h>
#include <geometry_msgs/Vector3.h>
#include "geometry_msgs/Quaternion.h"


using namespace std; 

// =========================================================================================================================
// This class file is to transport into include directory after development 

struct OctreeNode{
    float X; 
    float Y; 
    float Z; 
    uint8_t R; 
    uint8_t G; 
    uint8_t B; 
    int8_t Occ;  
    OctreeNode *TNW; // lll
    OctreeNode *TNE; // llh
    OctreeNode *TSE; // lhl
    OctreeNode *TSW; // lhh
    OctreeNode *BNW; // hll
    OctreeNode *BNE; // hlh
    OctreeNode *BSE; // hhl
    OctreeNode *BSW; // hhh
};

OctreeNode *current;            // these are global varibale used to travel down the tree 
OctreeNode *parent; 
queue<OctreeNode*> nodes;

class Octree{

    OctreeNode *root; 

    void insertNode(float x, float y, float z, uint8_t r, uint8_t g, uint8_t b){
            
        if(root == NULL){
            // If an obstabce node. 
            OctreeNode *new_node = new OctreeNode; 
            
            new_node->X = x; 
            new_node->Y = y; 
            new_node->Z = z; 
            new_node->R = r; 
            new_node->G = g; 
            new_node->B = b; 
            new_node->Occ = 1; 
            new_node->TNW = NULL;
            new_node->TNE = NULL;
            new_node->TSE = NULL;
            new_node->TSW = NULL;
            new_node->BNW = NULL;
            new_node->BNE = NULL;
            new_node->BSE = NULL;
            new_node->BSW = NULL;
            
            root = new_node;
            return; 
            //cout<<"   exited at root"<<endl;   
        }else{
            //insert from the root
            current = root; 
            parent = root; 
            
            while(current!=NULL){
                if(current->X == x && current->Y == y && current->Z == z){
                    //node previously entered
                    //cout<<"Similar: "<<x<<","<<y<<","<<z<<"\t"<<current->X<<","<<current->Y<<","<<current->Z<<endl; 
                    current->R = (current->R + r)/2; 
                    current->G = (current->G + g)/2; 
                    current->B = (current->B + b)/2; 
                    if(current->Occ < 16){
                        current->Occ +=1; 
                    }
                    return; 
                }
                else if(current->X >x && current->Y > y && current->Z > z){
                    parent = current; 
                    if(current->TNW != NULL){
                        current = current->TNW; 
                    }else{
                        current = NULL; 
                    }                
                }else if(current->X >x && current->Y > y && current->Z <= z){
                    parent = current;
                    if(current->TNE != NULL){
                        current = current->TNE;
                    }else{
                        current = NULL; 
                    }      
                }else if(current->X >x && current->Y <= y && current->Z > z){
                    parent = current;
                    if(current->TSE != NULL){
                        current = current->TSE;
                    }else{
                        current = NULL; 
                    }      
                }else if(current->X >x && current->Y <= y && current->Z <= z){
                    parent = current;
                    if(current->TSW != NULL){
                        current = current->TSW; 
                    }else{
                        current = NULL; 
                    } 
                }else if(current->X <= x && current->Y > y && current->Z > z){
                    parent = current;
                    if(current->BNW){
                        current = current->BNW;
                    } else{
                        current = NULL; 
                    }    
                }else if(current->X <= x && current->Y > y && current->Z <= z){
                    parent = current;
                    if(current->BNE != NULL){
                        current = current->BNE; 
                    }else{
                        current = NULL; 
                    }    
                }else if(current->X <= x && current->Y <= y && current->Z > z){
                    parent = current;
                    if(current->BSE){
                        current = current->BSE; 
                    } else{
                        current = NULL; 
                    }   
                }else if(current->X <= x && current->Y <= y && current->Z <= z){
                    parent = current;
                    if(current->BSW){
                        current = current->BSW; 
                    } else{
                        current = NULL; 
                    }      
                }else{
                    //std::cout<<" search returned"<<endl; 
                    return; // similar nodes cannot be added into the Octree for now. 
                }     
            }
            //cout<<"   search for: "<<x<<","<<y<<","<<z<<" stopped at: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<endl;
            
            // now we have the final leaf node as prev_node 
            // create the new node as a new Octree Node. 
            OctreeNode *new_node = new OctreeNode; 
            new_node->X = x; 
            new_node->Y = y; 
            new_node->Z = z; 
            new_node->R = r; 
            new_node->G = g; 
            new_node->B = b; 
            new_node->Occ = 1; 
            new_node->TNW = NULL;
            new_node->TNE = NULL;
            new_node->TSE = NULL;
            new_node->TSW = NULL;
            new_node->BNW = NULL;
            new_node->BNE = NULL;
            new_node->BSE = NULL;
            new_node->BSW = NULL;
            
            if(parent->X >x && parent->Y > y && parent->Z > z){
                parent->TNW = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  TNW node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X > x && parent->Y > y && parent->Z <= z){
                parent->TNE = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  TNE node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X > x && parent->Y <= y && parent->Z > z){
                parent->TSE = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  TSE node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X > x && parent->Y <= y && parent->Z <= z){
                parent->TSW = new_node;
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  TSW node is: "<<x<<","<<y<<","<<z<<endl;  
            }else if(parent->X <= x && parent->Y > y && parent->Z > z){
                parent->BNW = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  BNW node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X <= x && parent->Y > y && parent->Z <= z){
                parent->BNE = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  BNE node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X <= x && parent->Y <= y && parent->Z > z){
                parent->BSE = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  BSE node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X <= x && parent->Y <= y && parent->Z <= z){
                parent->BSW = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  BSW node is: "<<x<<","<<y<<","<<z<<endl; 
            }else{
                return; // similar nodes cannot be added into the Octree for now. 
            }
        }
    }

    void insertNode_Free(float x, float y, float z){
        
        // If an obstabce node.
            
        if(root ==NULL){
            OctreeNode *new_node = new OctreeNode; 
            new_node->X = x; 
            new_node->Y = y; 
            new_node->Z = z; 
            new_node->R = 0; 
            new_node->G = 255; 
            new_node->B = 0; 
            new_node->Occ = -1; 
            new_node->TNW = NULL;
            new_node->TNE = NULL;
            new_node->TSE = NULL;
            new_node->TSW = NULL;
            new_node->BNW = NULL;
            new_node->BNE = NULL;
            new_node->BSE = NULL;
            new_node->BSW = NULL;
            
            root = new_node;
            //cout<<"   exited at root"<<endl;  
            return;  
        }else{
            //insert from the root
            current = root; 
            parent = root; 
            
            while(current!=NULL){
                if(current->X == x && current->Y == y && current->Z == z){
                    //node previously entered
                    //cout<<"Similar: "<<x<<","<<y<<","<<z<<"\t"<<current->X<<","<<current->Y<<","<<current->Z<<endl; 
                    if(current->Occ > -16){
                        current->Occ -=1; 
                    }
                    return; 
                }
                else if(current->X >x && current->Y > y && current->Z > z){
                    parent = current; 
                    if(current->TNW != NULL){
                        current = current->TNW; 
                    }else{
                        current = NULL; 
                    }                
                }else if(current->X >x && current->Y > y && current->Z <= z){
                    parent = current;
                    if(current->TNE != NULL){
                        current = current->TNE;
                    }else{
                        current = NULL; 
                    }      
                }else if(current->X >x && current->Y <= y && current->Z > z){
                    parent = current;
                    if(current->TSE != NULL){
                        current = current->TSE;
                    }else{
                        current = NULL; 
                    }      
                }else if(current->X >x && current->Y <= y && current->Z <= z){
                    parent = current;
                    if(current->TSW != NULL){
                        current = current->TSW; 
                    }else{
                        current = NULL; 
                    } 
                }else if(current->X <= x && current->Y > y && current->Z > z){
                    parent = current;
                    if(current->BNW){
                        current = current->BNW;
                    } else{
                        current = NULL; 
                    }    
                }else if(current->X <= x && current->Y > y && current->Z <= z){
                    parent = current;
                    if(current->BNE != NULL){
                        current = current->BNE; 
                    }else{
                        current = NULL; 
                    }    
                }else if(current->X <= x && current->Y <= y && current->Z > z){
                    parent = current;
                    if(current->BSE){
                        current = current->BSE; 
                    } else{
                        current = NULL; 
                    }   
                }else if(current->X <= x && current->Y <= y && current->Z <= z){
                    parent = current;
                    if(current->BSW){
                        current = current->BSW; 
                    } else{
                        current = NULL; 
                    }      
                }else{
                    //std::cout<<" search returned"<<endl; 
                    return; // similar nodes cannot be added into the Octree for now. 
                }     
            }
            //cout<<"   search for: "<<x<<","<<y<<","<<z<<" stopped at: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<endl;
            
            // now we have the final leaf node as prev_node 
            OctreeNode *new_node = new OctreeNode; 
            new_node->X = x; 
            new_node->Y = y; 
            new_node->Z = z; 
            new_node->R = 0; 
            new_node->G = 255; 
            new_node->B = 0; 
            new_node->Occ = -1; 
            new_node->TNW = NULL;
            new_node->TNE = NULL;
            new_node->TSE = NULL;
            new_node->TSW = NULL;
            new_node->BNW = NULL;
            new_node->BNE = NULL;
            new_node->BSE = NULL;
            new_node->BSW = NULL;
            
            if(parent->X >x && parent->Y > y && parent->Z > z){
                parent->TNW = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  TNW node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X > x && parent->Y > y && parent->Z <= z){
                parent->TNE = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  TNE node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X > x && parent->Y <= y && parent->Z > z){
                parent->TSE = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  TSE node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X > x && parent->Y <= y && parent->Z <= z){
                parent->TSW = new_node;
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  TSW node is: "<<x<<","<<y<<","<<z<<endl;  
            }else if(parent->X <= x && parent->Y > y && parent->Z > z){
                parent->BNW = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  BNW node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X <= x && parent->Y > y && parent->Z <= z){
                parent->BNE = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  BNE node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X <= x && parent->Y <= y && parent->Z > z){
                parent->BSE = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  BSE node is: "<<x<<","<<y<<","<<z<<endl; 
            }else if(parent->X <= x && parent->Y <= y && parent->Z <= z){
                parent->BSW = new_node; 
                //cout<<"parent is: "<<parent->X<<","<<parent->Y<<","<<parent->Z<<"  BSW node is: "<<x<<","<<y<<","<<z<<endl; 
            }else{
                return; // similar nodes cannot be added into the Octree for now. 
            }
        }
    }


    void inOrderTraverse(OctreeNode *node){
        if(node == NULL){
            return; 
        }
        inOrderTraverse(node->TNW);  
        inOrderTraverse(node->TNE); 
        inOrderTraverse(node->TSE); 
        inOrderTraverse(node->TSW); 
        inOrderTraverse(node->BNW); 
        inOrderTraverse(node->BNE); 
        inOrderTraverse(node->BSE); 
        inOrderTraverse(node->BSW); 
        std::cout<<"("<<node->X<<", "<<node->Y<<", "<<node->Z<<")\n";
    }

    void put_nodes_in_queue(OctreeNode *node){
        if(node == NULL){
            return; 
        }
        put_nodes_in_queue(node->TNW);  
        put_nodes_in_queue(node->TNE); 
        put_nodes_in_queue(node->TSE); 
        put_nodes_in_queue(node->TSW); 
        put_nodes_in_queue(node->BNW); 
        put_nodes_in_queue(node->BNE); 
        put_nodes_in_queue(node->BSE); 
        put_nodes_in_queue(node->BSW); 
        nodes.push(node); 
        //std::cout<<"pushing: "<<node<<std::endl; 
    }

    void postOrderDelete(){
        if(root == NULL){
            return; 
        }
        stack<OctreeNode*> node_stack; 
        stack<char> id_stack; 
        OctreeNode *current = new OctreeNode; 
        current = root; 
        node_stack.push(current); 

        while(!node_stack.empty()){
            if(current->TNW == NULL && current->TNE == NULL && current->TSE == NULL && current->TSW == NULL &&
                current->BNW == NULL && current->BNE == NULL && current->BSE == NULL && current->BSW == NULL){
                    // no children 
                    node_stack.pop(); 
                    if(!node_stack.empty()){
                        // not at root
                        current = node_stack.top(); 
                        if(id_stack.top() == '0'){
                            current->TNW = NULL; 
                            id_stack.pop(); 
                        }else if(id_stack.top() == '1'){
                            current->TNE = NULL; 
                            id_stack.pop(); 
                        }else if(id_stack.top() == '2'){
                            current->TSE = NULL; 
                            id_stack.pop(); 
                        }else if(id_stack.top() == '3'){
                            current->TSW = NULL; 
                            id_stack.pop();
                        }else if(id_stack.top() == '4'){
                            current->BNW = NULL; 
                            id_stack.pop();
                        }else if(id_stack.top() == '5'){
                            current->BNE = NULL; 
                            id_stack.pop();
                        }else if(id_stack.top() == '6'){
                            current->BSE = NULL; 
                            id_stack.pop();
                        }else{
                            current->BSW = NULL; 
                            id_stack.pop();
                        }
                }else{
                    // if we've come to the root
                    //std::cout<<"deleting: "<<root->Key<<endl; 
                    root = NULL; 

                }
            }else if(current->TNW != NULL){
                current = current->TNW; 
                node_stack.push(current); 
                id_stack.push('0');
            }else if(current->TNE != NULL){
                current = current->TNE; 
                node_stack.push(current); 
                id_stack.push('1');
            }else if(current->TSE != NULL){
                current = current->TSE; 
                node_stack.push(current); 
                id_stack.push('2');
            }else if(current->TSW != NULL){
                current = current->TSW; 
                node_stack.push(current); 
                id_stack.push('3');
            }else if(current->BNW != NULL){
                current = current->BNW; 
                node_stack.push(current); 
                id_stack.push('4');
            }else if(current->BNE != NULL){
                current = current->BNE; 
                node_stack.push(current); 
                id_stack.push('5');
            }else if(current->BSE != NULL){
                current = current->BSE; 
                node_stack.push(current); 
                id_stack.push('6');
            }else if(current->BSW != NULL){
                current = current->BSW; 
                node_stack.push(current); 
                id_stack.push('7');
            }else{}
        }
    }

    OctreeNode *searchNode(OctreeNode *node, float x, float y, float z){
        if(node == NULL){
            return NULL; 
        }else if((node->X == x) && (node->Y == y) && (node->Z == z)){
            return node; 
        }else if((node->X >= x) && (node->Y >= y) && (node->Z >= z)){
            return searchNode(node->TNW, x, y, z); 
        }else if((node->X >= x) && (node->Y >= y) && (node->Z <= z)){
            return searchNode(node->TNE, x, y, z); 
        }else if((node->X >= x) && (node->Y <= y) && (node->Z >= z)){
            return searchNode(node->TSE, x, y, z); 
        }else if((node->X >= x) && (node->Y <= y) && (node->Z <= z)){
            return searchNode(node->TSW, x, y, z); 
        }else if((node->X <= x) && (node->Y >= y) && (node->Z >= z)){
            return searchNode(node->BNW, x, y, z); 
        }else if((node->X <= x) && (node->Y >= y) && (node->Z <= z)){
            return searchNode(node->BNE, x, y, z); 
        }else if((node->X <= x) && (node->Y <= y) && (node->Z >= z)){
            return searchNode(node->BSE, x, y, z); 
        }else if(node->X <= x && node->Y <= y && node->Z <= z){
            return searchNode(node->BSW, x, y, z);
        }else{
            return node;   
        }
    }
    
    OctreeNode *findMinNode(){
        if(root == NULL){
            return NULL; 
        }
        //OctreeNode *current = new OctreeNode; 
        current = NULL; 
        current = root; 
        while(current->TNW != NULL || current->TNE != NULL || current->TSE != NULL || current->TSW != NULL 
            || current->BNW != NULL || current->BNE != NULL || current->BSE != NULL || current->BSW != NULL){
                // while the current node has any children
                if(current->TNW != NULL){
                    current = current->TNW;                        
                }else if(current->TNE != NULL){
                    current = current->TNE;                       
                }else if(current->TSE != NULL){
                    current = current->TSE;
                }else if(current->TSW != NULL){
                    current = current->TSW;          
                }else if(current->BNW != NULL){
                    current = current->BNW; 
                }else if(current->BNE != NULL){
                    current = current->BNE;                    
                }else if(current->BSE != NULL){
                    current = current->BSE;         
                }else if(current->BSW != NULL){
                    current = current->BSW;                      
                }else{
                    // No children. no case
                }
            }
        return current; 
    }
    
    OctreeNode *findMaxNode(){
        if(root == NULL){
            return NULL; 
        }else{
            OctreeNode *current = new OctreeNode; 
            current = root; 
            while(current->TNW != NULL || current->TNE != NULL || current->TSE != NULL || current->TSW != NULL 
                || current->BNW != NULL || current->BNE != NULL || current->BSE != NULL || current->BSW != NULL){
                // while the current node has any children
                if(current->BSW != NULL){
                    current = current->BSW;                        
                }else if(current->BSE != NULL){
                    current = current->BSE;                       
                }else if(current->BNE != NULL){
                    current = current->BNE;
                }else if(current->BNW != NULL){
                    current = current->BNW;          
                }else if(current->TSW != NULL){
                    current = current->TSW; 
                }else if(current->TSE != NULL){
                    current = current->TSE;                    
                }else if(current->TNE != NULL){
                    current = current->TNE;         
                }else if(current->TNW != NULL){
                    current = current->TNW;                      
                }else{
                    // No children. no case
                }
            }
            return current; 
        }
    }
        void destroy(OctreeNode *root){
            if(root == NULL){
                return;
            }else{
                destroy(root->TNW);
                destroy(root->TNE);
                destroy(root->TSE); 
                destroy(root->TSW); 
                destroy(root->BNW);
                destroy(root->BNE);
                destroy(root->BSE); 
                destroy(root->BSW); 
                delete root;
            }
        }  

    public:
        Octree(OctreeNode *node){
            root = node; 
        }

        Octree(float half_res){
            OctreeNode *root_node = new OctreeNode; 
            root_node->X = 0.025f; 
            root_node->Y = 0.025f;
            root_node->Z = 0.025f;
            root_node->R = 255;     // add yellow to the center 
            root_node->G = 255; 
            root_node->B = 0; 
            root_node->TNW = NULL;
            root_node->TNE = NULL; 
            root_node->TSE = NULL;
            root_node->TSW = NULL;
            root_node->BNW = NULL;
            root_node->BNE = NULL; 
            root_node->BSE = NULL;
            root_node->BSW = NULL;
            root = root_node; 
        }
        ~Octree(){
            //postOrderDelete(); 
            destroy(root); 
            std::cout<<"destroyed"<<endl; 
        }
        void insert(float x, float y, float z, uint8_t r, uint8_t g, uint8_t b){
            insertNode(x, y, z, r, g, b); 
        }

        void insert_free(float x, float y, float z){
            insertNode_Free(x, y, z); 
        }
        void display(){
            inOrderTraverse(root); 
            std::cout<<std::endl; 
        }

        void put_in_queue(){
            put_nodes_in_queue(root); 
        }

        void searchNode(float x, float y, float z){
            root = searchNode(root, x, y, z); 
        }
        OctreeNode *begin(){
            return findMinNode(); 
        }
        OctreeNode *end(){
            return findMaxNode(); 
        }
        
        OctreeNode *getRoot(){
            return root; 
        }

        OctreeNode &getRoot_reference(){
            return *root; // return root as a memory reference. 
        }     
};

const float epsilon = 0.0125; // epsilon to compare two floats. this value depends on the resolution we consider. (resolution /4 or lower )

// ==============================================================================================================================

//using namespace std; 

#define THREADS_PER_BLOCK 256		// the optimal value is number of cuda cores, if (#cuda cores < max th.pr.blck). 256 for TX2
static int NUM_OF_BLOCKS = 1; 

__device__ const float resolution = 0.05f; 	// Resolution of 5 cm
__device__ const float half_resolution = resolution/2; // the half of the resolution. this value is used in calculations 
 
const float max_sensor_radius = 3.00f; 	// scan data further than this modulus value will not be considered. 
 
__global__ void round_off_positional_coords(float* x, float* y, float* z, float* x_result, float* y_result, float* z_result, 
	float x_trans, float y_trans, float z_trans, double sin_a, double sin_b, double sin_g, double cos_a, double cos_b, double cos_g){
	
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	
	 
	if( (fabs(x[index]) < max_sensor_radius) and (fabs(y[index]) < max_sensor_radius) and (fabs(z[index]) < max_sensor_radius)){
				
		//B: Transformation model 1 -----------yaw only----------- for flat surface navigation-----------------------------------
		/*float x_temp = x[index]*cos_a - y[index]*sin_a + x_trans;  
		float y_temp = x[index]*sin_a + y[index]*cos_a + y_trans; 
		float z_temp = z[index] + z_trans; 
		
		x_result[index] = (ceilf(x_temp / resolution))*0.05 - half_resolution;
		y_result[index] = (ceilf(y_temp / resolution))*0.05 - half_resolution; 
		z_result[index] = (ceilf(z_temp / resolution))*0.05 - half_resolution; */
		//------------------------------------------------------------------------------------------------
		
		// C: Transformation model 2 ---------- Roll, pitch, yaw combined--------- for inclined planes navigation --------------
		float x_temp = x[index]*cos_a*cos_b + y[index]*cos_a*sin_b*sin_g - y[index]*sin_a*cos_g + z[index]*cos_a*sin_b*cos_g + z[index]*sin_a*sin_g + x_trans;  
		float y_temp = x[index]*sin_a*cos_b + y[index]*sin_a*sin_b*sin_g + y[index]*cos_a*cos_g + z[index]*sin_a*sin_b*cos_g - z[index]*cos_a*sin_g + y_trans; 
		float z_temp = x[index]*sin_b*-1 + y[index]*cos_b*sin_g + z[index]*cos_b*cos_g + z_trans; 
		
		x_result[index] = (ceilf(x_temp / resolution))*0.05 - half_resolution;
		y_result[index] = (ceilf(y_temp / resolution))*0.05 - half_resolution; 
		z_result[index] = (ceilf(z_temp / resolution))*0.05 - half_resolution;

		// -----------------------------------------------------------------------------------------------------------------------------------------
		
	}else{
		x_result[index] = 0.00f; 
		y_result[index] = 0.00f; 
		z_result[index] = 0.00f; 

	} 
	
}

/*
void serial_remove_null_points(int camera_array_size, float* x, float* y, float* z, uint8_t* r, uint8_t* g, uint8_t* b,
													float* x_cropped, float* y_cropped, float* z_cropped, uint8_t* r_cropped, uint8_t* g_cropped, uint8_t* b_cropped){
	// x, y, z are input arrays with rounded and range controlled poitn clouds
	// x_cropped etc. stores the new array with 0,0,0 points removed. 
	int index = 0; 
	for(int i=0; i<camera_array_size; i++){			// size of original set of point cloud
		if(x[i] != 0.00f){
			x_cropped[index] = x[i]; 
			y_cropped[index] = y[i]; 
			z_cropped[index] = z[i]; 
			r_cropped[index] = r[i]; 
			g_cropped[index] = g[i]; 
			b_cropped[index] = b[i]; 
			
			index +=1; 
		}
	}
}
*/

Octree tree(half_resolution);  
/*
void serial_downsample_point_cloud(int array_size, float* x_cleaned, float* y_cleaned, float* z_cleaned, uint8_t* r_cleaned, uint8_t* g_cleaned, uint8_t* b_cleaned){

	Key3D temp_key; 
	VoxelBody temp_body; 
	
	for(int i=0; i<array_size; i++){
		temp_key = Key3D(x_cleaned[i], y_cleaned[i], z_cleaned[i]);
		temp_body = VoxelBody(r_cleaned[i], g_cleaned[i], b_cleaned[i]); 
		// Keep inerting for now. will genaralize rgb values and count later. 
		//voxel_map_downsampled.insert(std::make_pair<Key3D, VoxelBody>(temp_key, temp_body));
		std::map<Key3D, VoxelBody, VoxelKeyComparator>::iterator it; 
		it = voxel_map_downsampled.find(temp_key); 
		if(it !=voxel_map_downsampled.end()){
			it->second.updateVoxel(r_cleaned[i], g_cleaned[i], b_cleaned[i]); 
		}else{
			voxel_map_downsampled.insert(std::make_pair<Key3D, VoxelBody>(temp_key, temp_body));
		}
	}
	
}*/

int cudamain(sensor_msgs::PointCloud2 point_cloud_std, nav_msgs::Odometry odom_message_std, int size){ 

    double starttotal, endtotal; 
	starttotal = clock();
	
	//make_range_array(resolution, max_sensor_radius); 
	int array_size = size; 	 
    
    double start1, end1; 
	start1 = clock();
    //end1 = clock();
	//double time1 = (double)(end1 - start1);
    
    // convert quaternion orientation into roll, pitch, yaw representation 
	//double roll, pitch, yaw;
	double roll, pitch, yaw; 
	tf::Quaternion quat;
    tf::quaternionMsgToTF(odom_message_std.pose.pose.orientation, quat);	
    tf::Matrix3x3(quat).getRPY(roll, pitch, yaw);
	float x_position = (float) odom_message_std.pose.pose.position.x; 
	float y_position = (float) odom_message_std.pose.pose.position.y; 
	float z_position = (float) odom_message_std.pose.pose.position.z; 

	double sin_gamma = sin(roll); 
	double sin_beta = sin(pitch); 
	double sin_alpha = sin(yaw); 
	double cos_gamma = cos(roll); 
	double cos_beta = cos(pitch); 
	double cos_alpha = cos(yaw); 

	
	//std::cout<<"alpha: "<<yaw<<"  sin alpha: "<<sin_alpha<<std::endl;
	//std::cout<<"\n\n"; 

	int counter = 0; 
	int effective_point_count = 0; 
	//declare the arrray sets before reading the point cloud values 
	
	float *x, *y, *z; // for allocating position values of the points 
	float *x_rounded, *y_rounded, *z_rounded; 							// the intermediate results after rounding off the x, y, z, original values to the resolution 
	u_int8_t *r, *g, *b; // for color values of the point cloud 
	

	int size_position = array_size * sizeof(float);
	int size_color = array_size * sizeof(u_int8_t);
	
	x = (float *)malloc( size_position );
   	y = (float *)malloc( size_position );
	z = (float *)malloc( size_position );

	x_rounded = (float *)malloc( size_position );
   	y_rounded = (float *)malloc( size_position );
	z_rounded = (float *)malloc( size_position );

	r = (u_int8_t *)malloc( size_color );
    g = (u_int8_t *)malloc( size_color );
	b = (u_int8_t *)malloc( size_color );
	
	// positional data vector generation 
	for(sensor_msgs::PointCloud2ConstIterator<float> it(point_cloud_std, "x"); it!=it.end(); ++it){
		y[counter] = it[0] * -1; 
		z[counter] = it[1] * -1;
		x[counter] = it[2];
		counter+=1;  
		
	}
	counter = 0; 
    for(sensor_msgs::PointCloud2ConstIterator<u_int8_t> it_color(point_cloud_std, "rgb"); it_color!=it_color.end(); ++it_color){
		b[counter] = unsigned(it_color[0]);	
		g[counter] = unsigned(it_color[1]);	
		r[counter] = unsigned(it_color[2]); 
		counter+=1; 
	}
	counter = 0; 
    
    // double start1, end1; 
	// start1 = clock();
    end1 = clock();
	double time1 = (double)(end1 - start1);
    
	// Adjust the number of blocks to be a whole number. 
	NUM_OF_BLOCKS = (array_size + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK; 


	//The cuda device variables 
	float *d_x, *d_y, *d_z;
	float *d_x_rounded, *d_y_rounded, *d_z_rounded; 
	
	hipMalloc( (void **) &d_x, size_position);
	hipMalloc( (void **) &d_y, size_position);
	hipMalloc( (void **) &d_z, size_position);
	
	hipMalloc( (void **) &d_x_rounded, size_position);
	hipMalloc( (void **) &d_y_rounded, size_position);
	hipMalloc( (void **) &d_z_rounded, size_position);

    hipMemcpy( d_x, x, size_position, hipMemcpyHostToDevice );
	hipMemcpy( d_y, y, size_position, hipMemcpyHostToDevice );
	hipMemcpy( d_z, z, size_position, hipMemcpyHostToDevice );
	
	// GPU process START---------------------------------------------------------------------------------------------------------------------------------------------------------------
	//---------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	double start2, end2; 
	start2 = clock();
    //end1 = clock();
	//double time1 = (double)(end1 - start1);
    
	
	round_off_positional_coords<<<NUM_OF_BLOCKS, THREADS_PER_BLOCK>>>(d_x, d_y, d_z, d_x_rounded, d_y_rounded, d_z_rounded,
									x_position, y_position, z_position, sin_alpha, sin_beta, sin_gamma, cos_alpha, cos_beta, cos_gamma);
	
    // double start1, end1; 
    // start1 = clock();
    end2 = clock();
    double time2 = (double)(end2 - start2);
                                    
	hipMemcpy( x_rounded, d_x_rounded, size_position, hipMemcpyDeviceToHost );
	hipMemcpy( y_rounded, d_y_rounded, size_position, hipMemcpyDeviceToHost );
	hipMemcpy( z_rounded, d_z_rounded, size_position, hipMemcpyDeviceToHost );

	
	// GPU process END----------------------------------------------------------------------------------------------------------------------------------------------------------------
	//---------------------------------------------------------------------------------------------------------------------------------------------------------------------------------
	
	// check the output of this array
	
	//serial_downsample_point_cloud(array_size, x_rounded, y_rounded, z_rounded, r, g, b); 
    
    current = new OctreeNode;   //initiating OctreeNodes
    parent = new OctreeNode; 
    //Octree tree(half_resolution);               // create the new tree for every scan instance 
    
    double start3, end3; 
	start3 = clock();
    //end1 = clock();
	//double time1 = (double)(end1 - start1);
    
    for(int j=0; j<array_size; j++){
        if(x_rounded[j] != 0.00f){
            //cout<<"["<<j<<"] :  "<<x_rounded[j]<<","<<y_rounded[j]<<","<<z_rounded[j]<<"\t"<<unsigned(r[j])<<","<<unsigned(g[j])<<","<<unsigned(b[j])<<std::endl; 
            tree.insert(x_rounded[j], y_rounded[j], z_rounded[j], r[j], g[j], b[j]); 
        }
    }
    // double start3, end3; 
	// start3 = clock();
    end3 = clock();
	double time3 = (double)(end3 - start3);
    
    double start4, end4; 
	start4 = clock();
    //end1 = clock();
	//double time1 = (double)(end1 - start1);
    
	
    tree.put_in_queue(); 
    //std::cout<<"finished putting in queue"<<std::endl; 
    
    // while(!nodes.empty()){
    //     nodes.pop(); 
    // }
    // std::cout<<"finished clearing the queue"<<endl; 
    
    std::string map_file_name = "octree_nodes.txt";
	ofstream offile;
	offile.open(map_file_name.c_str(), ios::trunc);
	if(offile.is_open()) { 
		while(!nodes.empty()){
            current = nodes.front(); 
            //std::cout<<unsigned(current->Occ)<<endl; 
            offile<<current->X<<"\t"<<current->Y<<"\t"<<current->Z<<"\t"<<unsigned(current->R)<<"\t"<<unsigned(current->G)<<"\t"<<unsigned(current->B)<<std::endl;  
            nodes.pop(); 
        }				
	}
	//std::cout<<"file written"<<endl; 
	offile.close();
    // double start4, end4; 
	// start1 = clock();
    end4 = clock();
	double time4 = (double)(end4 - start4);
    
    
	free(x);
    free(y);
	free(z);
	free(r);
	free(g);
	free(b);
	free(x_rounded); 
	free(y_rounded); 
	free(z_rounded);
	
	hipFree( d_x );
	hipFree( d_y );
	hipFree( d_z );
	hipFree(d_x_rounded); 
	hipFree(d_y_rounded); 
	hipFree(d_z_rounded); 
    
    // double start1, end1; 
	// start1 = clock();
    endtotal = clock();
    double timetotal = (double)(endtotal - starttotal);
    
    std::cout<<time1<<"\t"<<time2<<"\t"<<time3<<"\t"<<time4<<"\t"<<timetotal<<endl; 

	return EXIT_SUCCESS; 	
}
