// This is for simple addition of two integers using cuda 
#include <stdio.h>

#include <hip/hip_runtime.h>



__global__ void add(int a, int b, int *c){
	*c = a+b; 
}

int testmain(void){
	int c; 
	int *dev_c;
	hipMalloc((void**)&dev_c, sizeof(int)); 
	
	add<<<1,1>>>(2,7,dev_c); 
		
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost); 
	
	printf("2+7 = %d\n", c); 
	hipFree(dev_c);
	return 0;  
}
