#include <stdio.h>
#include <time.h> 
#include <hip/hip_runtime.h>



__global__ void add(int *a, int *b, int *c, int n) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index <n){
		c[index] = a[index] + b[index];
	}
}


#define N (2048*2048)
#define THREADS_PER_BLOCK 512


int testmain(void){
	
	int *a, *b, *c; 
	int *d_a, *d_b, *d_c; 
	int size = N*sizeof(int); 
	
	//Allocate space for device copies a, b, c
	hipMalloc((void **)&d_a, size); 
	hipMalloc((void **)&d_b, size); 
	hipMalloc((void **)&d_c, size); 
	
	//Allocate space for host copies of a, b, c, and setup input values
	a = (int *)malloc(size); //random_ints(a, N); 
	b = (int *)malloc(size); //random_ints(b, N); 
	c = (int *)malloc(size);
	
	//Initialize two array in a simple way
	for(int i=0; i<N; i++){
		a[i] = -i; 
		b[i] = i*i; 
	}
	
	//Copy inputs to device 
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice); 
	
	
	//Launch add kernel on GPU with N blocks
	add<<<(N + THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N); 
	
	
	//Copy results back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	
	//clean up
	free(a); 
	free(b); 
	free(c);
	
	hipFree(d_a); 
	hipFree(d_b); 
	hipFree(d_c);
	
	return 0;
}


